#include "hip/hip_runtime.h"
﻿#include "hipblas.h"
#include "hip/hip_runtime.h"
#include "memory.h"
#include <iostream>
#include <ctime>

using namespace std;

typedef float real__t; //增强程序的可移植性
typedef unsigned int uint__t;

const int thread_num = 256;
const int block_num = 48;
const int blocksize = 1024*1024*48;

void select(real__t *A,long long n,long long k);

int CorMat_gpu(real__t * Cormat, real__t * BOLD, int N, int L, int Batch_size)
{
	real__t * BOLD_t1, * BOLD_t2, * out, * tempout;
	int Num_Blocks = (N + Batch_size - 1) / Batch_size;
	uint__t N0 = Num_Blocks * Batch_size;

	// transposing the BOLD signal
	real__t * BOLD_t = new real__t [L * N0];
	tempout = new real__t[Batch_size * Batch_size];
	memset(BOLD_t, 0, sizeof(real__t) * L * N0);
	for (int i = 0; i < L; i ++)
		for (int j = 0; j < N; j++)
		{
			BOLD_t[j * L + i] = BOLD[i * N + j];
		}

		// Normalize
		for (int i = 0; i < N; i++)
		{
			real__t * row = BOLD_t + i * L;
			double sum1 = 0, sum2 = 0;
			for (int l = 0; l < L; l++)
			{
				sum1 += row[l];
			}
			sum1 /= L;
			for (int l = 0; l < L; l++)
			{
				sum2 += (row[l] - sum1) * (row[l] - sum1);
			}
			sum2 = sqrt(sum2);
			for (int l = 0; l < L; l++)
			{
				row[l] = (row[l] - sum1) / sum2;;
			}
		}

		//// column major in every block
		//real__t * BOLD_t_col = new real__t [L * N0];
		//for (int k = 0; k < Num_Blocks; k++)
		//{
		//	for (int i = 0; i < Batch_size; i ++)
		//		for (int j = 0; j < L; j++)
		//		{
		//			BOLD_t_col[k * Batch_size * L + j * Batch_size + i] = BOLD_t[k * Batch_size * L + i * L + j];
		//		}
		//}

		hipError_t cudaStat;
		hipblasStatus_t stat;
		hipblasHandle_t handle;
		real__t * devBOLD, * devCormat;
//		stat = cublasAlloc(L*N0, sizeof(real__t), (void**)&devBOLD);
		cudaStat = hipMalloc ((void**)&devBOLD, sizeof(real__t) * L * N0) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
//		stat = cublasAlloc(Batch_size * Batch_size, sizeof(real__t), (void**)&devCormat);		
		cudaStat = hipMalloc ( (void**)&devCormat, sizeof(real__t) * Batch_size * Batch_size) ;
		if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
		stat = hipblasSetMatrix(N0, L, sizeof(real__t), BOLD_t, N0, devBOLD, N0);
//		cudaStat = hipMemcpy(devBOLD, BOLD_t, sizeof(real__t) * L * N0, hipMemcpyHostToDevice);
		stat = hipblasCreate(&handle) ;
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;

		const float alpha = 1.0;
		const float beta = 0;
		for (int kk = 0, ii = 0; ii < Num_Blocks; ii++)
			for (int jj = ii; jj < Num_Blocks; jj++)
			{
				BOLD_t1 = BOLD_t + ii * Batch_size * L;
				BOLD_t2 = BOLD_t + jj * Batch_size * L;
				out = Cormat + kk * Batch_size * Batch_size;
				kk++;
				stat = hipblasSgemm(handle, HIPBLAS_OP_T,  HIPBLAS_OP_N, Batch_size, Batch_size, L,  &alpha, devBOLD + jj * Batch_size * L, L, devBOLD + ii * Batch_size * L, L, &beta, devCormat, Batch_size);

				if (stat != HIPBLAS_STATUS_SUCCESS)
					return stat;

				cudaStat = hipMemcpy(out, devCormat, sizeof(real__t) * Batch_size * Batch_size, hipMemcpyDeviceToHost);
				if (cudaStat != hipSuccess) 
					return cudaStat;
				
		/*		for (int i = 0; i < Batch_size; i ++)
					for (int j = 0; j < Batch_size; j++)
					{
						out[j * Batch_size + i] = tempout[i * Batch_size + j];
					}*/
				/*float *testA = new float[Batch_size * L];
				hipMemcpy(testA, devBOLD + ii * Batch_size * L, sizeof(real__t) * Batch_size * L, hipMemcpyDeviceToHost);
				float *testB = new float[Batch_size * L];
				hipMemcpy(testB, devBOLD + jj * Batch_size * L, sizeof(real__t) * Batch_size * L, hipMemcpyDeviceToHost);
				cout<<"A"<<endl;
				for (int i = 0; i < Batch_size; i++)
				{
				for (int j = 0; j < L; j++)
				{
				cout<<testA[i * L + j]<<"\t";
				}
				cout<<endl;
				}
				getchar();
				cout<<"B"<<endl;

				for (int i = 0; i < Batch_size; i++)
				{
				for (int j = 0; j < L; j++)
				{
				cout<<testB[i * L + j]<<"\t";
				}
				cout<<endl;
				}
				getchar();
				cout<<"C"<<endl;
				for (int i = 0; i < Batch_size; i++)
				{
				for (int j = 0; j < Batch_size; j++)
				{
				cout<<out[i * Batch_size + j]<<"\t";
				}
				cout<<endl;
				}
				getchar();*/
				/*	double sum3;
				for (int k = 0, i = 0; i < Batch_size; i++)
					for (int j = 0; j < Batch_size; j++)
					{
						sum3 = 0;
						for (int l = 0; l < L; l++)
						{
							sum3 += BOLD_t1[i*L+l] * BOLD_t2[j*L+l];
						}
						out[k++] = sum3;
					}*/
			}
			hipFree (devBOLD); 
			hipFree (devCormat);
			stat = hipblasDestroy(handle);
			if (stat != HIPBLAS_STATUS_SUCCESS)
				return stat;
			delete []BOLD_t;
			return 1;
}

__global__ void partition_kernel ( real__t x, real__t *A,  double *j_l, double *j_r,const long N)
{
	__shared__ int tmp_j_l[thread_num];
	__shared__ int tmp_j_r[thread_num];
	int tmp_l = 0;
	int tmp_r = 0;
	int offset = 0;
	const int threadid =blockIdx.x*blockDim.x + threadIdx.x;

	for(offset=threadid; offset<N; offset+=blockDim.x*gridDim.x)
	{		
		if(A[offset]>x)
			tmp_l++ ;
		else if(A[offset]<x)
			tmp_r++;		
	}
	tmp_j_l[threadIdx.x] = tmp_l;
	tmp_j_r[threadIdx.x] = tmp_r;

	syncthreads();

	for(int i=1; i < thread_num; i*=2)
	{
			if (threadIdx.x%(2*i)==0)  
			{	
				tmp_j_l[threadIdx.x]+= tmp_j_l[threadIdx.x+i] ;
				tmp_j_r[threadIdx.x]+= tmp_j_r[threadIdx.x+i] ;
			}
			syncthreads();
	}	
	if(threadIdx.x==0)
		j_l[blockIdx.x]=(double) tmp_j_l[0];
	if(threadIdx.x==1)
		j_r[blockIdx.x]=(double) tmp_j_r[0];
}



/*int *partition_gpu()
{}*/
real__t select_GPU(real__t *Cormat, long long M1, long long k)
{
	long long offset;

	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	
	int index,i=0;
	//long long k_next = 0, M_next = 0;
	//int onethread_n = blocksize/block_num/thread_num;


	real__t *Cormat_block;
	cudaStat = hipMalloc ((void**)&Cormat_block, sizeof(real__t) * blocksize) ;
	if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
			
	int blocksize_num = (M1-1+blocksize)/blocksize;
	double *j_l;
	hipMalloc ((void**)&j_l, sizeof(double) * block_num*blocksize_num) ;
	double *j_r;
	hipMalloc ((void**)&j_r, sizeof(double) * block_num*blocksize_num) ;
	double hj_l = 0;
	double hj_r = 0; 
	//double *hj_ll = new double [block_num*blocksize_num];
	//double *hj_rr = new double [block_num*blocksize_num];
	
	real__t left  = 0.0;
	real__t right = 1.0;
	real__t x;
	stat = hipblasCreate(&handle) ;
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
	
	
	
	clock_t time = clock();
	while(true)
	{
		x = (left+right)/2.0;
		i=0;
		for (offset = 0; offset < M1; offset += blocksize)
		{
			int size = (M1-offset > blocksize? blocksize : M1-offset);
			hipblasSetVector(size, sizeof(real__t), Cormat+offset, 1, Cormat_block, 1);
			partition_kernel<<<block_num,thread_num>>>(x, Cormat_block,  j_l+block_num*i, j_r+block_num*i,(long) size);		
			i++;
		}
	//hipMemcpy(hj_ll, j_l, sizeof(double)*block_num*blocksize_num, hipMemcpyDeviceToHost);
	//hipMemcpy(hj_rr, j_r, sizeof(double)*block_num*blocksize_num, hipMemcpyDeviceToHost);

		stat = hipblasDasum(handle, block_num*blocksize_num, j_l, 1, &hj_l);
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;
		stat = hipblasDasum(handle, block_num*blocksize_num, j_r, 1, &hj_r);
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;
		//hipMemcpy(hj_r, j_r, sizeof(int)*block_num*((M1-1)/blocksize), hipMemcpyDeviceToHost);
		cout<<x<<endl;	
		cout<<"hj_l = "<<hj_l<<" ; hj_r = "<<hj_r<<";  k = "<<k<<endl;			
	
		if( (long long)(hj_l + hj_r) > M1) cout<<"partition error! hj_l = "<<hj_l<<" ; hj_r = "<<hj_r<<endl;
		else if ((long long)hj_l<=k && (long long)hj_r<=M1-k) break;
		else if (hj_l < k)  right = x;	
		else left = x;
	}
			
	time = clock() - time;
	cout<<"partition time = "<<time<<";  hj_l = "<<(long long)hj_l<<" ; hj_r = "<<(long long)hj_r<<endl;
	cout<<"r_threshold = "<<x<<endl;
	return (x);
	
}






long long find_max(real__t *Cormat, long long M1)
{
	long long offset;

	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	long long blocksize = 1024*1024*48;
	int index,i=0;

	real__t *Cormat_block;
	cudaStat = hipMalloc ((void**)&Cormat_block, sizeof(real__t) * blocksize) ;
	if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
			return cudaStat;
	
	stat = hipblasCreate(&handle) ;
		if (stat != HIPBLAS_STATUS_SUCCESS)
			return stat;

	int segnum = (M1+blocksize)/blocksize;
	real__t *tmp = new real__t [segnum];
	
	long long *tmp_index = new long long [segnum] ;
	

	for (offset = 0; offset + blocksize < M1; offset += blocksize)
	{	
		hipblasSetVector(blocksize, sizeof(real__t), Cormat+offset, 1, Cormat_block, 1);
		stat = hipblasIsamax(handle, blocksize, Cormat_block, 1, &index);
		tmp[i] = *(Cormat+offset+index-1);
		//cout << tmp[i]<<endl;
		tmp_index[i++] = index+offset-1;
		//cout<< tmp_index[i-1]<<endl;
	}
	hipblasSetVector(M1-offset, sizeof(real__t), Cormat+offset, 1, Cormat_block, 1);
	stat = hipblasIsamax(handle, M1-offset, Cormat_block, 1, &index);
	tmp[i] = *(Cormat+offset+index);
	tmp_index[i] = index+offset;
	//cout << tmp[i]<<endl;
	//cout<< tmp_index[i]<<endl;

	//real__t *tmp_gpu;
	//cudaStat = hipMalloc ((void**)&tmp_gpu, sizeof(real__t) * segnum) ;
	//if (cudaStat != HIPBLAS_STATUS_SUCCESS) 
	//		return cudaStat;
	
	//hipblasSetVector(segnum, sizeof(real__t), tmp, 1, tmp_gpu, 1);
	//stat = hipblasIsamax(handle, segnum, tmp_gpu, 1, &index);
	real__t max_r = tmp[0];
	for (i = 1; i<segnum; i++)
		if (tmp[i] > max_r)
		{	max_r = tmp[i]; index = i;  }  

	return tmp_index[index];
}


	