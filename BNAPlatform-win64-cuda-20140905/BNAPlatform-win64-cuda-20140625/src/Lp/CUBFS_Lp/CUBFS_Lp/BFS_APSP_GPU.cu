#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hip/device_functions.h"
#include "BFS_APSP_GPU.cuh"
#define VIRTUAL_WARP 8

__global__ void APSP_BFS_node_kernel(int *r, int *c, int * dist, int numVertices, int numEdges, int offset_source) 
{
int offset_vertices = blockIdx.x * numVertices;
	int offset_edge = blockIdx.x * numEdges;

	for (int i = threadIdx.x; i < numVertices; i += blockDim.x)
	{
		dist[offset_vertices + i] = -1;
	}

	int edge_index = threadIdx.x % VIRTUAL_WARP;
	int vertice_index = threadIdx.x / VIRTUAL_WARP;
	int source = blockIdx.x + offset_source;
	if (source >= numVertices)
		return;
	__shared__ bool done;
	done = false;
	
	int level = 0;
	dist[offset_vertices + source] = level++;
	while (!done)
	{
		__syncthreads(); // attention: this sync is neccessary
		done = true;
		for (int current = vertice_index; current < numVertices; current += blockDim.x / VIRTUAL_WARP)
		{
			if (dist[offset_vertices + current] != level - 1)
				continue;
			for (int j = r[current] + edge_index; j < r[current + 1]; j += VIRTUAL_WARP)
			{
				int next = c[j];
				int read_dist = dist[offset_vertices + next];
				if (read_dist == -1)
				{
					dist[offset_vertices + next] = level;
					done = false;
				}
			}
		}
		level ++;
			__syncthreads();  	
	}							  	
}

void APSP_GPU(int * dist, int *r, int *c, int numVertices, int numEdges, int grid, int thread)
{
    int devID;
    hipDeviceProp_t deviceProps;

    devID = findCudaDevice();

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    //printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

	//int thread = 256;
	//int grid = 100;
    // allocate device memory
    int* d_r; 
	int* d_c;
	int* d_dist;

    checkCudaErrors( hipMalloc( (void**) &d_r, sizeof(int) * (numVertices + 1)));
	checkCudaErrors( hipMalloc( (void**) &d_c, sizeof(int) * numEdges));

	
    // copy host memory to device
	checkCudaErrors( hipMemcpy( d_r, r, sizeof(int) * (numVertices + 1), hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy( d_c, c, sizeof(int) * numEdges, hipMemcpyHostToDevice) );

    // allocate device memory for result
	checkCudaErrors( hipMalloc( (void**) &d_dist, sizeof(int) * numVertices * grid));

	clock_t kernel_time = 0;
	clock_t transfer_time = 0;

    // execute the kernel
	for (int offset_source = 0; offset_source < numVertices; offset_source += grid)
	{
		clock_t time = clock();
		APSP_BFS_node_kernel<<<grid, thread>>>(d_r, d_c, d_dist, numVertices, numEdges, offset_source);
		// check if kernel execution generated and error
		getLastCudaError("Kernel execution failed");
		hipDeviceSynchronize();
		time = clock() - time;

		cout<<offset_source<<" done. Time = "<<time<<"ms."<<endl;
		kernel_time += time;

		time = clock();
		// copy result from device to host
		if(numVertices - offset_source > grid)			 
			checkCudaErrors(hipMemcpy(dist + (long long)offset_source * numVertices, d_dist, sizeof(float) * numVertices * grid, hipMemcpyDeviceToHost));
		else
			checkCudaErrors(hipMemcpy(dist + (long long) offset_source * numVertices, d_dist, sizeof(float) * numVertices * (numVertices%grid), hipMemcpyDeviceToHost));
		time = clock() - time;
		transfer_time += time;
	}

	cout<<"total kernel time: "<<kernel_time<<"ms."<<endl;
	cout<<"total transfering time: "<<transfer_time<<"ms."<<endl;

    // cleanup memory
    checkCudaErrors(hipFree(d_r));
    checkCudaErrors(hipFree(d_c));
	checkCudaErrors(hipFree(d_dist));
    hipDeviceReset();
}