#include "hip/hip_runtime.h"
# include <iostream>
# include "mytimer.h"
using namespace std;

# define SIZE_ROW_PER_THREAD_RX3 6
# define SIZE_COL_PER_THREAD_RX3 6
const int SHARED_BLOCK_SIZE = SIZE_ROW_PER_THREAD_RX3 * 16;

__global__ void cuBFW_ph2(float * dst_ij, float * src_ik, float * src_kj);

void BFW_CUDA_ph2(float * dst_ij, float * src_ik, float * src_kj, int block_size)
{
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuBFW_ph2), hipFuncCachePreferShared);
	dim3 dimBlock(16, 16); 
	dim3 dimGrid(block_size / SHARED_BLOCK_SIZE, block_size / SHARED_BLOCK_SIZE);

	STimer tmr;
	TimerInit(&tmr);
	TimerStart(&tmr);	

	cuBFW_ph2<<<dimGrid, dimBlock>>>(dst_ij, src_ik, src_kj);

/*	hipDeviceSynchronize();
	TimerStop(&tmr);
	double n1 = block_size / 1024.;
	double Gflop = 2*n1*n1*n1;
	cout<<"Phase 2 time: "<<TimerGetRuntime(&tmr)*1000<<"ms, Achieving: "<< Gflop / TimerGetRuntime(&tmr)<<"Gflop/s"<<endl;	
	*/
}

# define SHARED_FETCH_SIZE 16

__global__ void cuBFW_ph2(float * dst_ij, float * src_ik, float * src_kj)
{
	
	int DIM = SIZE_COL_PER_THREAD_RX3 * blockDim.x * gridDim.x;
	dst_ij += blockIdx.y * SHARED_BLOCK_SIZE * DIM + blockIdx.x * SHARED_BLOCK_SIZE;
	dst_ij += threadIdx.y * SIZE_ROW_PER_THREAD_RX3 * DIM + threadIdx.x * SIZE_COL_PER_THREAD_RX3;
	src_ik += blockIdx.y * SHARED_BLOCK_SIZE * DIM;
	src_kj += blockIdx.x * SHARED_BLOCK_SIZE;

	__shared__ float shared_src_kj1[SHARED_BLOCK_SIZE * SHARED_FETCH_SIZE]; 
	__shared__ float shared_src_ik1[SHARED_BLOCK_SIZE * SHARED_FETCH_SIZE]; 

	float reg_dst_ij_00, reg_dst_ij_01, reg_dst_ij_02, reg_dst_ij_03, reg_dst_ij_04, reg_dst_ij_05;
	float reg_dst_ij_10, reg_dst_ij_11, reg_dst_ij_12, reg_dst_ij_13, reg_dst_ij_14, reg_dst_ij_15;
	float reg_dst_ij_20, reg_dst_ij_21, reg_dst_ij_22, reg_dst_ij_23, reg_dst_ij_24, reg_dst_ij_25;
	float reg_dst_ij_30, reg_dst_ij_31, reg_dst_ij_32, reg_dst_ij_33, reg_dst_ij_34, reg_dst_ij_35;
	float reg_dst_ij_40, reg_dst_ij_41, reg_dst_ij_42, reg_dst_ij_43, reg_dst_ij_44, reg_dst_ij_45;
	float reg_dst_ij_50, reg_dst_ij_51, reg_dst_ij_52, reg_dst_ij_53, reg_dst_ij_54, reg_dst_ij_55;

	float reg_src_ik_0_, reg_src_ik_1_, reg_src_ik_2_, reg_src_ik_3_, reg_src_ik_4_, reg_src_ik_5_;
	float reg_src_kj__0, reg_src_kj__1, reg_src_kj__2, reg_src_kj__3, reg_src_kj__4, reg_src_kj__5;

	reg_dst_ij_00 = 1e10;//dst_ij[0 * DIM + 0];
	reg_dst_ij_01 = 1e10;//dst_ij[0 * DIM + 1];
	reg_dst_ij_02 = 1e10;//dst_ij[0 * DIM + 2];
	reg_dst_ij_03 = 1e10;//dst_ij[0 * DIM + 3];
	reg_dst_ij_04 = 1e10;//dst_ij[0 * DIM + 4];
	reg_dst_ij_05 = 1e10;//dst_ij[0 * DIM + 5];

	reg_dst_ij_10 = 1e10;//dst_ij[1 * DIM + 0];
	reg_dst_ij_11 = 1e10;//dst_ij[1 * DIM + 1];
	reg_dst_ij_12 = 1e10;//dst_ij[1 * DIM + 2];
	reg_dst_ij_13 = 1e10;//dst_ij[1 * DIM + 3];
	reg_dst_ij_14 = 1e10;//dst_ij[1 * DIM + 4];
	reg_dst_ij_15 = 1e10;//dst_ij[1 * DIM + 5];

	reg_dst_ij_20 = 1e10;//dst_ij[2 * DIM + 0];
	reg_dst_ij_21 = 1e10;//dst_ij[2 * DIM + 1];
	reg_dst_ij_22 = 1e10;//dst_ij[2 * DIM + 2];
	reg_dst_ij_23 = 1e10;//dst_ij[2 * DIM + 3];
	reg_dst_ij_24 = 1e10;//dst_ij[2 * DIM + 4];
	reg_dst_ij_25 = 1e10;//dst_ij[2 * DIM + 5];

	reg_dst_ij_30 = 1e10;//dst_ij[3 * DIM + 0];
	reg_dst_ij_31 = 1e10;//dst_ij[3 * DIM + 1];
	reg_dst_ij_32 = 1e10;//dst_ij[3 * DIM + 2];
	reg_dst_ij_33 = 1e10;//dst_ij[3 * DIM + 3];
	reg_dst_ij_34 = 1e10;//dst_ij[3 * DIM + 4];
	reg_dst_ij_35 = 1e10;//dst_ij[3 * DIM + 5];

	reg_dst_ij_40 = 1e10;//dst_ij[4 * DIM + 0];
	reg_dst_ij_41 = 1e10;//dst_ij[4 * DIM + 1];
	reg_dst_ij_42 = 1e10;//dst_ij[4 * DIM + 2];
	reg_dst_ij_43 = 1e10;//dst_ij[4 * DIM + 3];
	reg_dst_ij_44 = 1e10;//dst_ij[4 * DIM + 4];
	reg_dst_ij_45 = 1e10;//dst_ij[4 * DIM + 5];

	reg_dst_ij_50 = 1e10;//dst_ij[5 * DIM + 0];
	reg_dst_ij_51 = 1e10;//dst_ij[5 * DIM + 1];
	reg_dst_ij_52 = 1e10;//dst_ij[5 * DIM + 2];
	reg_dst_ij_53 = 1e10;//dst_ij[5 * DIM + 3];
	reg_dst_ij_54 = 1e10;//dst_ij[5 * DIM + 4];
	reg_dst_ij_55 = 1e10;//dst_ij[5 * DIM + 5];

	//int ID = threadIdx.y * blockDim.x + threadIdx.x;
	float * shared_src_ik_calc = shared_src_ik1 + threadIdx.y * SIZE_ROW_PER_THREAD_RX3 * SHARED_FETCH_SIZE;
	float * shared_src_kj_calc = shared_src_kj1 + threadIdx.x * SIZE_COL_PER_THREAD_RX3;

//	# pragma unroll 2
	for (int kk = 0; kk < DIM; kk += SHARED_FETCH_SIZE)
	{
		__syncthreads();
		shared_src_ik1[threadIdx.y * SHARED_FETCH_SIZE + threadIdx.x + 0 * 256] = src_ik[kk + threadIdx.y * DIM + threadIdx.x + 0 * 16 * DIM];
		shared_src_ik1[threadIdx.y * SHARED_FETCH_SIZE + threadIdx.x + 1 * 256] = src_ik[kk + threadIdx.y * DIM + threadIdx.x + 1 * 16 * DIM];
		shared_src_ik1[threadIdx.y * SHARED_FETCH_SIZE + threadIdx.x + 2 * 256] = src_ik[kk + threadIdx.y * DIM + threadIdx.x + 2 * 16 * DIM];
		shared_src_ik1[threadIdx.y * SHARED_FETCH_SIZE + threadIdx.x + 3 * 256] = src_ik[kk + threadIdx.y * DIM + threadIdx.x + 3 * 16 * DIM];
		shared_src_ik1[threadIdx.y * SHARED_FETCH_SIZE + threadIdx.x + 4 * 256] = src_ik[kk + threadIdx.y * DIM + threadIdx.x + 4 * 16 * DIM];
		shared_src_ik1[threadIdx.y * SHARED_FETCH_SIZE + threadIdx.x + 5 * 256] = src_ik[kk + threadIdx.y * DIM + threadIdx.x + 5 * 16 * DIM];

		shared_src_kj1[threadIdx.y * SHARED_BLOCK_SIZE + threadIdx.x + 0 * 16] = src_kj[kk * DIM + threadIdx.y * DIM + threadIdx.x + 0 * 16];
		shared_src_kj1[threadIdx.y * SHARED_BLOCK_SIZE + threadIdx.x + 1 * 16] = src_kj[kk * DIM + threadIdx.y * DIM + threadIdx.x + 1 * 16];
		shared_src_kj1[threadIdx.y * SHARED_BLOCK_SIZE + threadIdx.x + 2 * 16] = src_kj[kk * DIM + threadIdx.y * DIM + threadIdx.x + 2 * 16];
		shared_src_kj1[threadIdx.y * SHARED_BLOCK_SIZE + threadIdx.x + 3 * 16] = src_kj[kk * DIM + threadIdx.y * DIM + threadIdx.x + 3 * 16];
		shared_src_kj1[threadIdx.y * SHARED_BLOCK_SIZE + threadIdx.x + 4 * 16] = src_kj[kk * DIM + threadIdx.y * DIM + threadIdx.x + 4 * 16];
		shared_src_kj1[threadIdx.y * SHARED_BLOCK_SIZE + threadIdx.x + 5 * 16] = src_kj[kk * DIM + threadIdx.y * DIM + threadIdx.x + 5 * 16];

		__syncthreads();

		//# pragma unroll 2
		for (int k = 0; k < SHARED_FETCH_SIZE; k++)
		{		
			reg_src_ik_0_ = shared_src_ik_calc[0 * SHARED_FETCH_SIZE + k];
			reg_src_ik_1_ = shared_src_ik_calc[1 * SHARED_FETCH_SIZE + k];
			reg_src_ik_2_ = shared_src_ik_calc[2 * SHARED_FETCH_SIZE + k];
			reg_src_ik_3_ = shared_src_ik_calc[3 * SHARED_FETCH_SIZE + k];
			reg_src_ik_4_ = shared_src_ik_calc[4 * SHARED_FETCH_SIZE + k];
			reg_src_ik_5_ = shared_src_ik_calc[5 * SHARED_FETCH_SIZE + k];
									 
			reg_src_kj__0 = shared_src_kj_calc[0 + k * SHARED_BLOCK_SIZE];
			reg_src_kj__1 = shared_src_kj_calc[1 + k * SHARED_BLOCK_SIZE];
			reg_src_kj__2 = shared_src_kj_calc[2 + k * SHARED_BLOCK_SIZE];
			reg_src_kj__3 = shared_src_kj_calc[3 + k * SHARED_BLOCK_SIZE];
			reg_src_kj__4 = shared_src_kj_calc[4 + k * SHARED_BLOCK_SIZE];
			reg_src_kj__5 = shared_src_kj_calc[5 + k * SHARED_BLOCK_SIZE];

			reg_dst_ij_00 = fminf(reg_dst_ij_00, reg_src_ik_0_ + reg_src_kj__0);
			reg_dst_ij_10 = fminf(reg_dst_ij_10, reg_src_ik_1_ + reg_src_kj__0);
			reg_dst_ij_20 = fminf(reg_dst_ij_20, reg_src_ik_2_ + reg_src_kj__0);
			reg_dst_ij_30 = fminf(reg_dst_ij_30, reg_src_ik_3_ + reg_src_kj__0);
			reg_dst_ij_40 = fminf(reg_dst_ij_40, reg_src_ik_4_ + reg_src_kj__0);
			reg_dst_ij_50 = fminf(reg_dst_ij_50, reg_src_ik_5_ + reg_src_kj__0);
																
			reg_dst_ij_01 = fminf(reg_dst_ij_01, reg_src_ik_0_ + reg_src_kj__1);
			reg_dst_ij_11 = fminf(reg_dst_ij_11, reg_src_ik_1_ + reg_src_kj__1);
			reg_dst_ij_21 = fminf(reg_dst_ij_21, reg_src_ik_2_ + reg_src_kj__1);
			reg_dst_ij_31 = fminf(reg_dst_ij_31, reg_src_ik_3_ + reg_src_kj__1);
			reg_dst_ij_41 = fminf(reg_dst_ij_41, reg_src_ik_4_ + reg_src_kj__1);
			reg_dst_ij_51 = fminf(reg_dst_ij_51, reg_src_ik_5_ + reg_src_kj__1);
																			
			reg_dst_ij_02 = fminf(reg_dst_ij_02, reg_src_ik_0_ + reg_src_kj__2);
			reg_dst_ij_12 = fminf(reg_dst_ij_12, reg_src_ik_1_ + reg_src_kj__2);
			reg_dst_ij_22 = fminf(reg_dst_ij_22, reg_src_ik_2_ + reg_src_kj__2);
			reg_dst_ij_32 = fminf(reg_dst_ij_32, reg_src_ik_3_ + reg_src_kj__2);
			reg_dst_ij_42 = fminf(reg_dst_ij_42, reg_src_ik_4_ + reg_src_kj__2);
			reg_dst_ij_52 = fminf(reg_dst_ij_52, reg_src_ik_5_ + reg_src_kj__2);
																			
			reg_dst_ij_03 = fminf(reg_dst_ij_03, reg_src_ik_0_ + reg_src_kj__3);
			reg_dst_ij_13 = fminf(reg_dst_ij_13, reg_src_ik_1_ + reg_src_kj__3);
			reg_dst_ij_23 = fminf(reg_dst_ij_23, reg_src_ik_2_ + reg_src_kj__3);
			reg_dst_ij_33 = fminf(reg_dst_ij_33, reg_src_ik_3_ + reg_src_kj__3);
			reg_dst_ij_43 = fminf(reg_dst_ij_43, reg_src_ik_4_ + reg_src_kj__3);
			reg_dst_ij_53 = fminf(reg_dst_ij_53, reg_src_ik_5_ + reg_src_kj__3);
																			
			reg_dst_ij_04 = fminf(reg_dst_ij_04, reg_src_ik_0_ + reg_src_kj__4);
			reg_dst_ij_14 = fminf(reg_dst_ij_14, reg_src_ik_1_ + reg_src_kj__4);
			reg_dst_ij_24 = fminf(reg_dst_ij_24, reg_src_ik_2_ + reg_src_kj__4);
			reg_dst_ij_34 = fminf(reg_dst_ij_34, reg_src_ik_3_ + reg_src_kj__4);
			reg_dst_ij_44 = fminf(reg_dst_ij_44, reg_src_ik_4_ + reg_src_kj__4);
			reg_dst_ij_54 = fminf(reg_dst_ij_54, reg_src_ik_5_ + reg_src_kj__4);
														   				
			reg_dst_ij_05 = fminf(reg_dst_ij_05, reg_src_ik_0_ + reg_src_kj__5);
			reg_dst_ij_15 = fminf(reg_dst_ij_15, reg_src_ik_1_ + reg_src_kj__5);
			reg_dst_ij_25 = fminf(reg_dst_ij_25, reg_src_ik_2_ + reg_src_kj__5);
			reg_dst_ij_35 = fminf(reg_dst_ij_35, reg_src_ik_3_ + reg_src_kj__5);
			reg_dst_ij_45 = fminf(reg_dst_ij_45, reg_src_ik_4_ + reg_src_kj__5);
			reg_dst_ij_55 = fminf(reg_dst_ij_55, reg_src_ik_5_ + reg_src_kj__5);
			
		}
	}

	dst_ij[0 * DIM + 0] = reg_dst_ij_00;
	dst_ij[0 * DIM + 1] = reg_dst_ij_01;
	dst_ij[0 * DIM + 2] = reg_dst_ij_02;
	dst_ij[0 * DIM + 3] = reg_dst_ij_03;
	dst_ij[0 * DIM + 4] = reg_dst_ij_04;
	dst_ij[0 * DIM + 5] = reg_dst_ij_05;

	dst_ij[1 * DIM + 0] = reg_dst_ij_10;
	dst_ij[1 * DIM + 1] = reg_dst_ij_11;
	dst_ij[1 * DIM + 2] = reg_dst_ij_12;
	dst_ij[1 * DIM + 3] = reg_dst_ij_13;
	dst_ij[1 * DIM + 4] = reg_dst_ij_14;
	dst_ij[1 * DIM + 5] = reg_dst_ij_15;

	dst_ij[2 * DIM + 0] = reg_dst_ij_20;
	dst_ij[2 * DIM + 1] = reg_dst_ij_21;
	dst_ij[2 * DIM + 2] = reg_dst_ij_22;
	dst_ij[2 * DIM + 3] = reg_dst_ij_23;
	dst_ij[2 * DIM + 4] = reg_dst_ij_24;
	dst_ij[2 * DIM + 5] = reg_dst_ij_25;

	dst_ij[3 * DIM + 0] = reg_dst_ij_30;
	dst_ij[3 * DIM + 1] = reg_dst_ij_31;
	dst_ij[3 * DIM + 2] = reg_dst_ij_32;
	dst_ij[3 * DIM + 3] = reg_dst_ij_33;
	dst_ij[3 * DIM + 4] = reg_dst_ij_34;
	dst_ij[3 * DIM + 5] = reg_dst_ij_35;

	dst_ij[4 * DIM + 0] = reg_dst_ij_40;
	dst_ij[4 * DIM + 1] = reg_dst_ij_41;
	dst_ij[4 * DIM + 2] = reg_dst_ij_42;
	dst_ij[4 * DIM + 3] = reg_dst_ij_43;
	dst_ij[4 * DIM + 4] = reg_dst_ij_44;
	dst_ij[4 * DIM + 5] = reg_dst_ij_45;

	dst_ij[5 * DIM + 0] = reg_dst_ij_50;
	dst_ij[5 * DIM + 1] = reg_dst_ij_51;
	dst_ij[5 * DIM + 2] = reg_dst_ij_52;
	dst_ij[5 * DIM + 3] = reg_dst_ij_53;
	dst_ij[5 * DIM + 4] = reg_dst_ij_54;
	dst_ij[5 * DIM + 5] = reg_dst_ij_55;

}