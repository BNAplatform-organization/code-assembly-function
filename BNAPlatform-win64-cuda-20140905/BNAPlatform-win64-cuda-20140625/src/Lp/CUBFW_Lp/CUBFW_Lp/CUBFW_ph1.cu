#include "hip/hip_runtime.h"
# include <iostream>
# include "mytimer.h"
# include "FormBlock.h"
using namespace std;

# define SIZE_ROW_PER_THREAD_KATZ 6
# define SIZE_COL_PER_THREAD_KATZ 6
const int Katz_Block_Size = SIZE_ROW_PER_THREAD_KATZ * 16;

# define SHARED_FETCH_SIZE 16
__global__ void cuBFW_small_Katz_ph123(float * dst_ij, int k_block, int phase);
__global__ void cuBFW_small_Katz_ph2(float * dst_ij, int k_block);
__global__ void cuBFW_small_Katz_ph3(float * dst_ij, int k_block);

#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
	if(hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
		exit(-1);        
	}
}

#define getLastCudaError(msg)      __getLastCudaError (msg, __FILE__, __LINE__)
inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
{
	hipError_t err = hipGetLastError();
	if (hipSuccess != err)
	{
		fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
			file, line, errorMessage, (int)err, hipGetErrorString( err ) );
		exit(-1);
	}
}

void BFW_CUDA_small_Katz(float *costmat, float * kernel_primary_block, int dim)
{
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuBFW_small_Katz_ph123), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuBFW_small_Katz_ph2), hipFuncCachePreferShared);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(cuBFW_small_Katz_ph3), hipFuncCachePreferShared);

	float * costmat_katz = new float[dim * dim];
	FormBlock(costmat_katz, costmat, dim, Katz_Block_Size);
	checkCudaErrors(hipMemcpy(kernel_primary_block, costmat_katz, sizeof(float) * dim * dim, hipMemcpyHostToDevice));

	int block_cnt = dim / Katz_Block_Size;

	dim3 dimBlock(Katz_Block_Size / SIZE_COL_PER_THREAD_KATZ, Katz_Block_Size / SIZE_ROW_PER_THREAD_KATZ); 
	dim3 dimGrid(block_cnt, block_cnt);

	STimer tmr;
	TimerInit(&tmr);
	TimerStart(&tmr);	
	for (int k_block = 0; k_block < block_cnt; k_block++)
	{
		cuBFW_small_Katz_ph123<<<dimGrid, dimBlock>>>(kernel_primary_block, k_block, 1);
		cuBFW_small_Katz_ph2<<<dimGrid, dimBlock>>>(kernel_primary_block, k_block);
		cuBFW_small_Katz_ph3<<<dimGrid, dimBlock>>>(kernel_primary_block, k_block);
	}
	getLastCudaError("Kernel execution failed");
/*	hipDeviceSynchronize();
	TimerStop(&tmr);

	double n1 = Katz_Block_Size / 1024.;
	double Gflop = 2*n1*n1*n1 * (block_cnt)* (block_cnt) * block_cnt;
	cout<<"Phase 1 time: "<<TimerGetRuntime(&tmr)*1000<<"ms, Achieving: "<< Gflop / TimerGetRuntime(&tmr)<<"Gflop/s"<<endl;	
*/
	checkCudaErrors(hipMemcpy(costmat_katz, kernel_primary_block, sizeof(float) * dim * dim, hipMemcpyDeviceToHost));
	DeFormBlock(costmat, costmat_katz, dim, Katz_Block_Size);
	delete []costmat_katz;
}

# define SHARED_FETCH_SIZE 16

__global__ void cuBFW_small_Katz_ph3(float * dst_ij, int k_block)
{
	if (blockIdx.y == k_block || blockIdx.x == k_block)
		return;
	float * src_ik = dst_ij + (blockIdx.y * gridDim.x + k_block) * Katz_Block_Size * Katz_Block_Size;
	float * src_kj = dst_ij + (k_block * gridDim.x + blockIdx.x) * Katz_Block_Size * Katz_Block_Size;
	dst_ij += (blockIdx.y * gridDim.x + blockIdx.x) * Katz_Block_Size * Katz_Block_Size;
	dst_ij += threadIdx.y * SIZE_ROW_PER_THREAD_KATZ * Katz_Block_Size + threadIdx.x * SIZE_COL_PER_THREAD_KATZ;
	int ID = threadIdx.y * blockDim.x + threadIdx.x;

	__shared__ float shared_src_kj1[Katz_Block_Size * SHARED_FETCH_SIZE]; 
	__shared__ float shared_src_ik1[Katz_Block_Size * SHARED_FETCH_SIZE]; 

	float * shared_src_ik_calc = shared_src_ik1 + threadIdx.y * SIZE_ROW_PER_THREAD_KATZ * SHARED_FETCH_SIZE;
	float * shared_src_kj_calc = shared_src_kj1 + threadIdx.x * SIZE_COL_PER_THREAD_KATZ;

	float reg_dst_ij_00, reg_dst_ij_01, reg_dst_ij_02, reg_dst_ij_03, reg_dst_ij_04, reg_dst_ij_05;
	float reg_dst_ij_10, reg_dst_ij_11, reg_dst_ij_12, reg_dst_ij_13, reg_dst_ij_14, reg_dst_ij_15;
	float reg_dst_ij_20, reg_dst_ij_21, reg_dst_ij_22, reg_dst_ij_23, reg_dst_ij_24, reg_dst_ij_25;
	float reg_dst_ij_30, reg_dst_ij_31, reg_dst_ij_32, reg_dst_ij_33, reg_dst_ij_34, reg_dst_ij_35;
	float reg_dst_ij_40, reg_dst_ij_41, reg_dst_ij_42, reg_dst_ij_43, reg_dst_ij_44, reg_dst_ij_45;
	float reg_dst_ij_50, reg_dst_ij_51, reg_dst_ij_52, reg_dst_ij_53, reg_dst_ij_54, reg_dst_ij_55;

	float reg_src_ik_0_, reg_src_ik_1_, reg_src_ik_2_, reg_src_ik_3_, reg_src_ik_4_, reg_src_ik_5_;
	float reg_src_kj__0, reg_src_kj__1, reg_src_kj__2, reg_src_kj__3, reg_src_kj__4, reg_src_kj__5;

	reg_dst_ij_00 = dst_ij[0 * Katz_Block_Size + 0];
	reg_dst_ij_01 = dst_ij[0 * Katz_Block_Size + 1];
	reg_dst_ij_02 = dst_ij[0 * Katz_Block_Size + 2];
	reg_dst_ij_03 = dst_ij[0 * Katz_Block_Size + 3];
	reg_dst_ij_04 = dst_ij[0 * Katz_Block_Size + 4];
	reg_dst_ij_05 = dst_ij[0 * Katz_Block_Size + 5];

	reg_dst_ij_10 = dst_ij[1 * Katz_Block_Size + 0];
	reg_dst_ij_11 = dst_ij[1 * Katz_Block_Size + 1];
	reg_dst_ij_12 = dst_ij[1 * Katz_Block_Size + 2];
	reg_dst_ij_13 = dst_ij[1 * Katz_Block_Size + 3];
	reg_dst_ij_14 = dst_ij[1 * Katz_Block_Size + 4];
	reg_dst_ij_15 = dst_ij[1 * Katz_Block_Size + 5];

	reg_dst_ij_20 = dst_ij[2 * Katz_Block_Size + 0];
	reg_dst_ij_21 = dst_ij[2 * Katz_Block_Size + 1];
	reg_dst_ij_22 = dst_ij[2 * Katz_Block_Size + 2];
	reg_dst_ij_23 = dst_ij[2 * Katz_Block_Size + 3];
	reg_dst_ij_24 = dst_ij[2 * Katz_Block_Size + 4];
	reg_dst_ij_25 = dst_ij[2 * Katz_Block_Size + 5];

	reg_dst_ij_30 = dst_ij[3 * Katz_Block_Size + 0];
	reg_dst_ij_31 = dst_ij[3 * Katz_Block_Size + 1];
	reg_dst_ij_32 = dst_ij[3 * Katz_Block_Size + 2];
	reg_dst_ij_33 = dst_ij[3 * Katz_Block_Size + 3];
	reg_dst_ij_34 = dst_ij[3 * Katz_Block_Size + 4];
	reg_dst_ij_35 = dst_ij[3 * Katz_Block_Size + 5];

	reg_dst_ij_40 = dst_ij[4 * Katz_Block_Size + 0];
	reg_dst_ij_41 = dst_ij[4 * Katz_Block_Size + 1];
	reg_dst_ij_42 = dst_ij[4 * Katz_Block_Size + 2];
	reg_dst_ij_43 = dst_ij[4 * Katz_Block_Size + 3];
	reg_dst_ij_44 = dst_ij[4 * Katz_Block_Size + 4];
	reg_dst_ij_45 = dst_ij[4 * Katz_Block_Size + 5];

	reg_dst_ij_50 = dst_ij[5 * Katz_Block_Size + 0];
	reg_dst_ij_51 = dst_ij[5 * Katz_Block_Size + 1];
	reg_dst_ij_52 = dst_ij[5 * Katz_Block_Size + 2];
	reg_dst_ij_53 = dst_ij[5 * Katz_Block_Size + 3];
	reg_dst_ij_54 = dst_ij[5 * Katz_Block_Size + 4];
	reg_dst_ij_55 = dst_ij[5 * Katz_Block_Size + 5];

//	# pragma unroll 2
	for (int kk = 0; kk < Katz_Block_Size; kk += SHARED_FETCH_SIZE)
	{
		__syncthreads();
		shared_src_ik1[ID + 0 * 256] = src_ik[kk + (threadIdx.y + 0 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 1 * 256] = src_ik[kk + (threadIdx.y + 1 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 2 * 256] = src_ik[kk + (threadIdx.y + 2 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 3 * 256] = src_ik[kk + (threadIdx.y + 3 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 4 * 256] = src_ik[kk + (threadIdx.y + 4 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 5 * 256] = src_ik[kk + (threadIdx.y + 5 * blockDim.y) * Katz_Block_Size + threadIdx.x];

		shared_src_kj1[ID + 0 * 256] = src_kj[kk * Katz_Block_Size + ID + 0 * 256];
		shared_src_kj1[ID + 1 * 256] = src_kj[kk * Katz_Block_Size + ID + 1 * 256];
		shared_src_kj1[ID + 2 * 256] = src_kj[kk * Katz_Block_Size + ID + 2 * 256];
		shared_src_kj1[ID + 3 * 256] = src_kj[kk * Katz_Block_Size + ID + 3 * 256];
		shared_src_kj1[ID + 4 * 256] = src_kj[kk * Katz_Block_Size + ID + 4 * 256];
		shared_src_kj1[ID + 5 * 256] = src_kj[kk * Katz_Block_Size + ID + 5 * 256];
		__syncthreads();

		//# pragma unroll 2
		for (int k = 0; k < SHARED_FETCH_SIZE; k++)
		{		
			reg_src_ik_0_ = shared_src_ik_calc[0 * SHARED_FETCH_SIZE + k];
			reg_src_ik_1_ = shared_src_ik_calc[1 * SHARED_FETCH_SIZE + k];
			reg_src_ik_2_ = shared_src_ik_calc[2 * SHARED_FETCH_SIZE + k];
			reg_src_ik_3_ = shared_src_ik_calc[3 * SHARED_FETCH_SIZE + k];
			reg_src_ik_4_ = shared_src_ik_calc[4 * SHARED_FETCH_SIZE + k];
			reg_src_ik_5_ = shared_src_ik_calc[5 * SHARED_FETCH_SIZE + k];
									 
			reg_src_kj__0 = shared_src_kj_calc[0 + k * Katz_Block_Size];
			reg_src_kj__1 = shared_src_kj_calc[1 + k * Katz_Block_Size];
			reg_src_kj__2 = shared_src_kj_calc[2 + k * Katz_Block_Size];
			reg_src_kj__3 = shared_src_kj_calc[3 + k * Katz_Block_Size];
			reg_src_kj__4 = shared_src_kj_calc[4 + k * Katz_Block_Size];
			reg_src_kj__5 = shared_src_kj_calc[5 + k * Katz_Block_Size];

			reg_dst_ij_00 = fminf(reg_dst_ij_00, reg_src_ik_0_ + reg_src_kj__0);
			reg_dst_ij_10 = fminf(reg_dst_ij_10, reg_src_ik_1_ + reg_src_kj__0);
			reg_dst_ij_20 = fminf(reg_dst_ij_20, reg_src_ik_2_ + reg_src_kj__0);
			reg_dst_ij_30 = fminf(reg_dst_ij_30, reg_src_ik_3_ + reg_src_kj__0);
			reg_dst_ij_40 = fminf(reg_dst_ij_40, reg_src_ik_4_ + reg_src_kj__0);
			reg_dst_ij_50 = fminf(reg_dst_ij_50, reg_src_ik_5_ + reg_src_kj__0);
																
			reg_dst_ij_01 = fminf(reg_dst_ij_01, reg_src_ik_0_ + reg_src_kj__1);
			reg_dst_ij_11 = fminf(reg_dst_ij_11, reg_src_ik_1_ + reg_src_kj__1);
			reg_dst_ij_21 = fminf(reg_dst_ij_21, reg_src_ik_2_ + reg_src_kj__1);
			reg_dst_ij_31 = fminf(reg_dst_ij_31, reg_src_ik_3_ + reg_src_kj__1);
			reg_dst_ij_41 = fminf(reg_dst_ij_41, reg_src_ik_4_ + reg_src_kj__1);
			reg_dst_ij_51 = fminf(reg_dst_ij_51, reg_src_ik_5_ + reg_src_kj__1);
																			
			reg_dst_ij_02 = fminf(reg_dst_ij_02, reg_src_ik_0_ + reg_src_kj__2);
			reg_dst_ij_12 = fminf(reg_dst_ij_12, reg_src_ik_1_ + reg_src_kj__2);
			reg_dst_ij_22 = fminf(reg_dst_ij_22, reg_src_ik_2_ + reg_src_kj__2);
			reg_dst_ij_32 = fminf(reg_dst_ij_32, reg_src_ik_3_ + reg_src_kj__2);
			reg_dst_ij_42 = fminf(reg_dst_ij_42, reg_src_ik_4_ + reg_src_kj__2);
			reg_dst_ij_52 = fminf(reg_dst_ij_52, reg_src_ik_5_ + reg_src_kj__2);
																			
			reg_dst_ij_03 = fminf(reg_dst_ij_03, reg_src_ik_0_ + reg_src_kj__3);
			reg_dst_ij_13 = fminf(reg_dst_ij_13, reg_src_ik_1_ + reg_src_kj__3);
			reg_dst_ij_23 = fminf(reg_dst_ij_23, reg_src_ik_2_ + reg_src_kj__3);
			reg_dst_ij_33 = fminf(reg_dst_ij_33, reg_src_ik_3_ + reg_src_kj__3);
			reg_dst_ij_43 = fminf(reg_dst_ij_43, reg_src_ik_4_ + reg_src_kj__3);
			reg_dst_ij_53 = fminf(reg_dst_ij_53, reg_src_ik_5_ + reg_src_kj__3);
																			
			reg_dst_ij_04 = fminf(reg_dst_ij_04, reg_src_ik_0_ + reg_src_kj__4);
			reg_dst_ij_14 = fminf(reg_dst_ij_14, reg_src_ik_1_ + reg_src_kj__4);
			reg_dst_ij_24 = fminf(reg_dst_ij_24, reg_src_ik_2_ + reg_src_kj__4);
			reg_dst_ij_34 = fminf(reg_dst_ij_34, reg_src_ik_3_ + reg_src_kj__4);
			reg_dst_ij_44 = fminf(reg_dst_ij_44, reg_src_ik_4_ + reg_src_kj__4);
			reg_dst_ij_54 = fminf(reg_dst_ij_54, reg_src_ik_5_ + reg_src_kj__4);
														   				
			reg_dst_ij_05 = fminf(reg_dst_ij_05, reg_src_ik_0_ + reg_src_kj__5);
			reg_dst_ij_15 = fminf(reg_dst_ij_15, reg_src_ik_1_ + reg_src_kj__5);
			reg_dst_ij_25 = fminf(reg_dst_ij_25, reg_src_ik_2_ + reg_src_kj__5);
			reg_dst_ij_35 = fminf(reg_dst_ij_35, reg_src_ik_3_ + reg_src_kj__5);
			reg_dst_ij_45 = fminf(reg_dst_ij_45, reg_src_ik_4_ + reg_src_kj__5);
			reg_dst_ij_55 = fminf(reg_dst_ij_55, reg_src_ik_5_ + reg_src_kj__5);
		}
	}

	dst_ij[0 * Katz_Block_Size + 0] = reg_dst_ij_00;
	dst_ij[0 * Katz_Block_Size + 1] = reg_dst_ij_01;
	dst_ij[0 * Katz_Block_Size + 2] = reg_dst_ij_02;
	dst_ij[0 * Katz_Block_Size + 3] = reg_dst_ij_03;
	dst_ij[0 * Katz_Block_Size + 4] = reg_dst_ij_04;
	dst_ij[0 * Katz_Block_Size + 5] = reg_dst_ij_05;

	dst_ij[1 * Katz_Block_Size + 0] = reg_dst_ij_10;
	dst_ij[1 * Katz_Block_Size + 1] = reg_dst_ij_11;
	dst_ij[1 * Katz_Block_Size + 2] = reg_dst_ij_12;
	dst_ij[1 * Katz_Block_Size + 3] = reg_dst_ij_13;
	dst_ij[1 * Katz_Block_Size + 4] = reg_dst_ij_14;
	dst_ij[1 * Katz_Block_Size + 5] = reg_dst_ij_15;

	dst_ij[2 * Katz_Block_Size + 0] = reg_dst_ij_20;
	dst_ij[2 * Katz_Block_Size + 1] = reg_dst_ij_21;
	dst_ij[2 * Katz_Block_Size + 2] = reg_dst_ij_22;
	dst_ij[2 * Katz_Block_Size + 3] = reg_dst_ij_23;
	dst_ij[2 * Katz_Block_Size + 4] = reg_dst_ij_24;
	dst_ij[2 * Katz_Block_Size + 5] = reg_dst_ij_25;

	dst_ij[3 * Katz_Block_Size + 0] = reg_dst_ij_30;
	dst_ij[3 * Katz_Block_Size + 1] = reg_dst_ij_31;
	dst_ij[3 * Katz_Block_Size + 2] = reg_dst_ij_32;
	dst_ij[3 * Katz_Block_Size + 3] = reg_dst_ij_33;
	dst_ij[3 * Katz_Block_Size + 4] = reg_dst_ij_34;
	dst_ij[3 * Katz_Block_Size + 5] = reg_dst_ij_35;

	dst_ij[4 * Katz_Block_Size + 0] = reg_dst_ij_40;
	dst_ij[4 * Katz_Block_Size + 1] = reg_dst_ij_41;
	dst_ij[4 * Katz_Block_Size + 2] = reg_dst_ij_42;
	dst_ij[4 * Katz_Block_Size + 3] = reg_dst_ij_43;
	dst_ij[4 * Katz_Block_Size + 4] = reg_dst_ij_44;
	dst_ij[4 * Katz_Block_Size + 5] = reg_dst_ij_45;

	dst_ij[5 * Katz_Block_Size + 0] = reg_dst_ij_50;
	dst_ij[5 * Katz_Block_Size + 1] = reg_dst_ij_51;
	dst_ij[5 * Katz_Block_Size + 2] = reg_dst_ij_52;
	dst_ij[5 * Katz_Block_Size + 3] = reg_dst_ij_53;
	dst_ij[5 * Katz_Block_Size + 4] = reg_dst_ij_54;
	dst_ij[5 * Katz_Block_Size + 5] = reg_dst_ij_55;
}

__global__ void cuBFW_small_Katz_ph2(float * dst_ij, int k_block)
{
	if ((blockIdx.y != k_block && blockIdx.x != k_block) || (blockIdx.y == k_block && blockIdx.x == k_block))
		return;
	float * src_ik = dst_ij + (blockIdx.y * gridDim.x + k_block) * Katz_Block_Size * Katz_Block_Size;
	float * src_kj = dst_ij + (k_block * gridDim.x + blockIdx.x) * Katz_Block_Size * Katz_Block_Size;
	dst_ij += (blockIdx.y * gridDim.x + blockIdx.x) * Katz_Block_Size * Katz_Block_Size;
	dst_ij += threadIdx.y * SIZE_ROW_PER_THREAD_KATZ * Katz_Block_Size + threadIdx.x * SIZE_COL_PER_THREAD_KATZ;
	int ID = threadIdx.y * blockDim.x + threadIdx.x;

	__shared__ float shared_src_kj1[Katz_Block_Size * SHARED_FETCH_SIZE]; 
	__shared__ float shared_src_ik1[Katz_Block_Size * SHARED_FETCH_SIZE]; 

	float * shared_src_ik_calc = shared_src_ik1 + threadIdx.y * SIZE_ROW_PER_THREAD_KATZ * SHARED_FETCH_SIZE;
	float * shared_src_kj_calc = shared_src_kj1 + threadIdx.x * SIZE_COL_PER_THREAD_KATZ;

	float reg_dst_ij_00, reg_dst_ij_01, reg_dst_ij_02, reg_dst_ij_03, reg_dst_ij_04, reg_dst_ij_05;
	float reg_dst_ij_10, reg_dst_ij_11, reg_dst_ij_12, reg_dst_ij_13, reg_dst_ij_14, reg_dst_ij_15;
	float reg_dst_ij_20, reg_dst_ij_21, reg_dst_ij_22, reg_dst_ij_23, reg_dst_ij_24, reg_dst_ij_25;
	float reg_dst_ij_30, reg_dst_ij_31, reg_dst_ij_32, reg_dst_ij_33, reg_dst_ij_34, reg_dst_ij_35;
	float reg_dst_ij_40, reg_dst_ij_41, reg_dst_ij_42, reg_dst_ij_43, reg_dst_ij_44, reg_dst_ij_45;
	float reg_dst_ij_50, reg_dst_ij_51, reg_dst_ij_52, reg_dst_ij_53, reg_dst_ij_54, reg_dst_ij_55;

	float reg_src_ik_0_, reg_src_ik_1_, reg_src_ik_2_, reg_src_ik_3_, reg_src_ik_4_, reg_src_ik_5_;
	float reg_src_kj__0, reg_src_kj__1, reg_src_kj__2, reg_src_kj__3, reg_src_kj__4, reg_src_kj__5;

	reg_dst_ij_00 = 1e10;//dst_ij[0 * Katz_Block_Size + 0];
	reg_dst_ij_01 = 1e10;//dst_ij[0 * Katz_Block_Size + 1];
	reg_dst_ij_02 = 1e10;//dst_ij[0 * Katz_Block_Size + 2];
	reg_dst_ij_03 = 1e10;//dst_ij[0 * Katz_Block_Size + 3];
	reg_dst_ij_04 = 1e10;//dst_ij[0 * Katz_Block_Size + 4];
	reg_dst_ij_05 = 1e10;//dst_ij[0 * Katz_Block_Size + 5];

	reg_dst_ij_10 = 1e10;//dst_ij[1 * Katz_Block_Size + 0];
	reg_dst_ij_11 = 1e10;//dst_ij[1 * Katz_Block_Size + 1];
	reg_dst_ij_12 = 1e10;//dst_ij[1 * Katz_Block_Size + 2];
	reg_dst_ij_13 = 1e10;//dst_ij[1 * Katz_Block_Size + 3];
	reg_dst_ij_14 = 1e10;//dst_ij[1 * Katz_Block_Size + 4];
	reg_dst_ij_15 = 1e10;//dst_ij[1 * Katz_Block_Size + 5];

	reg_dst_ij_20 = 1e10;//dst_ij[2 * Katz_Block_Size + 0];
	reg_dst_ij_21 = 1e10;//dst_ij[2 * Katz_Block_Size + 1];
	reg_dst_ij_22 = 1e10;//dst_ij[2 * Katz_Block_Size + 2];
	reg_dst_ij_23 = 1e10;//dst_ij[2 * Katz_Block_Size + 3];
	reg_dst_ij_24 = 1e10;//dst_ij[2 * Katz_Block_Size + 4];
	reg_dst_ij_25 = 1e10;//dst_ij[2 * Katz_Block_Size + 5];

	reg_dst_ij_30 = 1e10;//dst_ij[3 * Katz_Block_Size + 0];
	reg_dst_ij_31 = 1e10;//dst_ij[3 * Katz_Block_Size + 1];
	reg_dst_ij_32 = 1e10;//dst_ij[3 * Katz_Block_Size + 2];
	reg_dst_ij_33 = 1e10;//dst_ij[3 * Katz_Block_Size + 3];
	reg_dst_ij_34 = 1e10;//dst_ij[3 * Katz_Block_Size + 4];
	reg_dst_ij_35 = 1e10;//dst_ij[3 * Katz_Block_Size + 5];

	reg_dst_ij_40 = 1e10;//dst_ij[4 * Katz_Block_Size + 0];
	reg_dst_ij_41 = 1e10;//dst_ij[4 * Katz_Block_Size + 1];
	reg_dst_ij_42 = 1e10;//dst_ij[4 * Katz_Block_Size + 2];
	reg_dst_ij_43 = 1e10;//dst_ij[4 * Katz_Block_Size + 3];
	reg_dst_ij_44 = 1e10;//dst_ij[4 * Katz_Block_Size + 4];
	reg_dst_ij_45 = 1e10;//dst_ij[4 * Katz_Block_Size + 5];

	reg_dst_ij_50 = 1e10;//dst_ij[5 * Katz_Block_Size + 0];
	reg_dst_ij_51 = 1e10;//dst_ij[5 * Katz_Block_Size + 1];
	reg_dst_ij_52 = 1e10;//dst_ij[5 * Katz_Block_Size + 2];
	reg_dst_ij_53 = 1e10;//dst_ij[5 * Katz_Block_Size + 3];
	reg_dst_ij_54 = 1e10;//dst_ij[5 * Katz_Block_Size + 4];
	reg_dst_ij_55 = 1e10;//dst_ij[5 * Katz_Block_Size + 5];

//	# pragma unroll 2
	for (int kk = 0; kk < Katz_Block_Size; kk += SHARED_FETCH_SIZE)
	{
		__syncthreads();
		shared_src_ik1[ID + 0 * 256] = src_ik[kk + (threadIdx.y + 0 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 1 * 256] = src_ik[kk + (threadIdx.y + 1 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 2 * 256] = src_ik[kk + (threadIdx.y + 2 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 3 * 256] = src_ik[kk + (threadIdx.y + 3 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 4 * 256] = src_ik[kk + (threadIdx.y + 4 * blockDim.y) * Katz_Block_Size + threadIdx.x];
		shared_src_ik1[ID + 5 * 256] = src_ik[kk + (threadIdx.y + 5 * blockDim.y) * Katz_Block_Size + threadIdx.x];

		shared_src_kj1[ID + 0 * 256] = src_kj[kk * Katz_Block_Size + ID + 0 * 256];
		shared_src_kj1[ID + 1 * 256] = src_kj[kk * Katz_Block_Size + ID + 1 * 256];
		shared_src_kj1[ID + 2 * 256] = src_kj[kk * Katz_Block_Size + ID + 2 * 256];
		shared_src_kj1[ID + 3 * 256] = src_kj[kk * Katz_Block_Size + ID + 3 * 256];
		shared_src_kj1[ID + 4 * 256] = src_kj[kk * Katz_Block_Size + ID + 4 * 256];
		shared_src_kj1[ID + 5 * 256] = src_kj[kk * Katz_Block_Size + ID + 5 * 256];
		__syncthreads();

		//# pragma unroll 2
		for (int k = 0; k < SHARED_FETCH_SIZE; k++)
		{		
			reg_src_ik_0_ = shared_src_ik_calc[0 * SHARED_FETCH_SIZE + k];
			reg_src_ik_1_ = shared_src_ik_calc[1 * SHARED_FETCH_SIZE + k];
			reg_src_ik_2_ = shared_src_ik_calc[2 * SHARED_FETCH_SIZE + k];
			reg_src_ik_3_ = shared_src_ik_calc[3 * SHARED_FETCH_SIZE + k];
			reg_src_ik_4_ = shared_src_ik_calc[4 * SHARED_FETCH_SIZE + k];
			reg_src_ik_5_ = shared_src_ik_calc[5 * SHARED_FETCH_SIZE + k];
									 
			reg_src_kj__0 = shared_src_kj_calc[0 + k * Katz_Block_Size];
			reg_src_kj__1 = shared_src_kj_calc[1 + k * Katz_Block_Size];
			reg_src_kj__2 = shared_src_kj_calc[2 + k * Katz_Block_Size];
			reg_src_kj__3 = shared_src_kj_calc[3 + k * Katz_Block_Size];
			reg_src_kj__4 = shared_src_kj_calc[4 + k * Katz_Block_Size];
			reg_src_kj__5 = shared_src_kj_calc[5 + k * Katz_Block_Size];

			reg_dst_ij_00 = fminf(reg_dst_ij_00, reg_src_ik_0_ + reg_src_kj__0);
			reg_dst_ij_10 = fminf(reg_dst_ij_10, reg_src_ik_1_ + reg_src_kj__0);
			reg_dst_ij_20 = fminf(reg_dst_ij_20, reg_src_ik_2_ + reg_src_kj__0);
			reg_dst_ij_30 = fminf(reg_dst_ij_30, reg_src_ik_3_ + reg_src_kj__0);
			reg_dst_ij_40 = fminf(reg_dst_ij_40, reg_src_ik_4_ + reg_src_kj__0);
			reg_dst_ij_50 = fminf(reg_dst_ij_50, reg_src_ik_5_ + reg_src_kj__0);
																
			reg_dst_ij_01 = fminf(reg_dst_ij_01, reg_src_ik_0_ + reg_src_kj__1);
			reg_dst_ij_11 = fminf(reg_dst_ij_11, reg_src_ik_1_ + reg_src_kj__1);
			reg_dst_ij_21 = fminf(reg_dst_ij_21, reg_src_ik_2_ + reg_src_kj__1);
			reg_dst_ij_31 = fminf(reg_dst_ij_31, reg_src_ik_3_ + reg_src_kj__1);
			reg_dst_ij_41 = fminf(reg_dst_ij_41, reg_src_ik_4_ + reg_src_kj__1);
			reg_dst_ij_51 = fminf(reg_dst_ij_51, reg_src_ik_5_ + reg_src_kj__1);
																			
			reg_dst_ij_02 = fminf(reg_dst_ij_02, reg_src_ik_0_ + reg_src_kj__2);
			reg_dst_ij_12 = fminf(reg_dst_ij_12, reg_src_ik_1_ + reg_src_kj__2);
			reg_dst_ij_22 = fminf(reg_dst_ij_22, reg_src_ik_2_ + reg_src_kj__2);
			reg_dst_ij_32 = fminf(reg_dst_ij_32, reg_src_ik_3_ + reg_src_kj__2);
			reg_dst_ij_42 = fminf(reg_dst_ij_42, reg_src_ik_4_ + reg_src_kj__2);
			reg_dst_ij_52 = fminf(reg_dst_ij_52, reg_src_ik_5_ + reg_src_kj__2);
																			
			reg_dst_ij_03 = fminf(reg_dst_ij_03, reg_src_ik_0_ + reg_src_kj__3);
			reg_dst_ij_13 = fminf(reg_dst_ij_13, reg_src_ik_1_ + reg_src_kj__3);
			reg_dst_ij_23 = fminf(reg_dst_ij_23, reg_src_ik_2_ + reg_src_kj__3);
			reg_dst_ij_33 = fminf(reg_dst_ij_33, reg_src_ik_3_ + reg_src_kj__3);
			reg_dst_ij_43 = fminf(reg_dst_ij_43, reg_src_ik_4_ + reg_src_kj__3);
			reg_dst_ij_53 = fminf(reg_dst_ij_53, reg_src_ik_5_ + reg_src_kj__3);
																			
			reg_dst_ij_04 = fminf(reg_dst_ij_04, reg_src_ik_0_ + reg_src_kj__4);
			reg_dst_ij_14 = fminf(reg_dst_ij_14, reg_src_ik_1_ + reg_src_kj__4);
			reg_dst_ij_24 = fminf(reg_dst_ij_24, reg_src_ik_2_ + reg_src_kj__4);
			reg_dst_ij_34 = fminf(reg_dst_ij_34, reg_src_ik_3_ + reg_src_kj__4);
			reg_dst_ij_44 = fminf(reg_dst_ij_44, reg_src_ik_4_ + reg_src_kj__4);
			reg_dst_ij_54 = fminf(reg_dst_ij_54, reg_src_ik_5_ + reg_src_kj__4);
														   				
			reg_dst_ij_05 = fminf(reg_dst_ij_05, reg_src_ik_0_ + reg_src_kj__5);
			reg_dst_ij_15 = fminf(reg_dst_ij_15, reg_src_ik_1_ + reg_src_kj__5);
			reg_dst_ij_25 = fminf(reg_dst_ij_25, reg_src_ik_2_ + reg_src_kj__5);
			reg_dst_ij_35 = fminf(reg_dst_ij_35, reg_src_ik_3_ + reg_src_kj__5);
			reg_dst_ij_45 = fminf(reg_dst_ij_45, reg_src_ik_4_ + reg_src_kj__5);
			reg_dst_ij_55 = fminf(reg_dst_ij_55, reg_src_ik_5_ + reg_src_kj__5);
		}
	}

	dst_ij[0 * Katz_Block_Size + 0] = reg_dst_ij_00;
	dst_ij[0 * Katz_Block_Size + 1] = reg_dst_ij_01;
	dst_ij[0 * Katz_Block_Size + 2] = reg_dst_ij_02;
	dst_ij[0 * Katz_Block_Size + 3] = reg_dst_ij_03;
	dst_ij[0 * Katz_Block_Size + 4] = reg_dst_ij_04;
	dst_ij[0 * Katz_Block_Size + 5] = reg_dst_ij_05;

	dst_ij[1 * Katz_Block_Size + 0] = reg_dst_ij_10;
	dst_ij[1 * Katz_Block_Size + 1] = reg_dst_ij_11;
	dst_ij[1 * Katz_Block_Size + 2] = reg_dst_ij_12;
	dst_ij[1 * Katz_Block_Size + 3] = reg_dst_ij_13;
	dst_ij[1 * Katz_Block_Size + 4] = reg_dst_ij_14;
	dst_ij[1 * Katz_Block_Size + 5] = reg_dst_ij_15;

	dst_ij[2 * Katz_Block_Size + 0] = reg_dst_ij_20;
	dst_ij[2 * Katz_Block_Size + 1] = reg_dst_ij_21;
	dst_ij[2 * Katz_Block_Size + 2] = reg_dst_ij_22;
	dst_ij[2 * Katz_Block_Size + 3] = reg_dst_ij_23;
	dst_ij[2 * Katz_Block_Size + 4] = reg_dst_ij_24;
	dst_ij[2 * Katz_Block_Size + 5] = reg_dst_ij_25;

	dst_ij[3 * Katz_Block_Size + 0] = reg_dst_ij_30;
	dst_ij[3 * Katz_Block_Size + 1] = reg_dst_ij_31;
	dst_ij[3 * Katz_Block_Size + 2] = reg_dst_ij_32;
	dst_ij[3 * Katz_Block_Size + 3] = reg_dst_ij_33;
	dst_ij[3 * Katz_Block_Size + 4] = reg_dst_ij_34;
	dst_ij[3 * Katz_Block_Size + 5] = reg_dst_ij_35;

	dst_ij[4 * Katz_Block_Size + 0] = reg_dst_ij_40;
	dst_ij[4 * Katz_Block_Size + 1] = reg_dst_ij_41;
	dst_ij[4 * Katz_Block_Size + 2] = reg_dst_ij_42;
	dst_ij[4 * Katz_Block_Size + 3] = reg_dst_ij_43;
	dst_ij[4 * Katz_Block_Size + 4] = reg_dst_ij_44;
	dst_ij[4 * Katz_Block_Size + 5] = reg_dst_ij_45;

	dst_ij[5 * Katz_Block_Size + 0] = reg_dst_ij_50;
	dst_ij[5 * Katz_Block_Size + 1] = reg_dst_ij_51;
	dst_ij[5 * Katz_Block_Size + 2] = reg_dst_ij_52;
	dst_ij[5 * Katz_Block_Size + 3] = reg_dst_ij_53;
	dst_ij[5 * Katz_Block_Size + 4] = reg_dst_ij_54;
	dst_ij[5 * Katz_Block_Size + 5] = reg_dst_ij_55;
}

// no shared memory usage
__global__ void cuBFW_small_Katz_ph123(float * dst_ij, int k_block, int phase)
{
	if (phase == 3 && (blockIdx.y == k_block || blockIdx.x == k_block))
	{
		return;
	}
	if (phase == 2 && ((blockIdx.y != k_block && blockIdx.x != k_block) || (blockIdx.y == k_block && blockIdx.x == k_block)))
	{
		return;
	}
	if (phase == 1 && (blockIdx.y != k_block || blockIdx.x != k_block))
	{
		return;
	}

	float * src_ik = dst_ij + (blockIdx.y * gridDim.x + k_block) * Katz_Block_Size * Katz_Block_Size;
	float * src_kj = dst_ij + (k_block * gridDim.x + blockIdx.x) * Katz_Block_Size * Katz_Block_Size;
	dst_ij += (blockIdx.y * gridDim.x + blockIdx.x) * Katz_Block_Size * Katz_Block_Size;
	dst_ij += threadIdx.y * SIZE_ROW_PER_THREAD_KATZ * Katz_Block_Size + threadIdx.x * SIZE_COL_PER_THREAD_KATZ;
	src_kj += threadIdx.x * SIZE_COL_PER_THREAD_KATZ;
	src_ik += threadIdx.y * SIZE_ROW_PER_THREAD_KATZ * Katz_Block_Size;	

	float reg_dst_ij_00, reg_dst_ij_01, reg_dst_ij_02, reg_dst_ij_03, reg_dst_ij_04, reg_dst_ij_05;
	float reg_dst_ij_10, reg_dst_ij_11, reg_dst_ij_12, reg_dst_ij_13, reg_dst_ij_14, reg_dst_ij_15;
	float reg_dst_ij_20, reg_dst_ij_21, reg_dst_ij_22, reg_dst_ij_23, reg_dst_ij_24, reg_dst_ij_25;
	float reg_dst_ij_30, reg_dst_ij_31, reg_dst_ij_32, reg_dst_ij_33, reg_dst_ij_34, reg_dst_ij_35;
	float reg_dst_ij_40, reg_dst_ij_41, reg_dst_ij_42, reg_dst_ij_43, reg_dst_ij_44, reg_dst_ij_45;
	float reg_dst_ij_50, reg_dst_ij_51, reg_dst_ij_52, reg_dst_ij_53, reg_dst_ij_54, reg_dst_ij_55;

	float reg_src_ik_0_, reg_src_ik_1_, reg_src_ik_2_, reg_src_ik_3_, reg_src_ik_4_, reg_src_ik_5_;
	float reg_src_kj__0, reg_src_kj__1, reg_src_kj__2, reg_src_kj__3, reg_src_kj__4, reg_src_kj__5;

	reg_dst_ij_00 = dst_ij[0 * Katz_Block_Size + 0];
	reg_dst_ij_01 = dst_ij[0 * Katz_Block_Size + 1];
	reg_dst_ij_02 = dst_ij[0 * Katz_Block_Size + 2];
	reg_dst_ij_03 = dst_ij[0 * Katz_Block_Size + 3];
	reg_dst_ij_04 = dst_ij[0 * Katz_Block_Size + 4];
	reg_dst_ij_05 = dst_ij[0 * Katz_Block_Size + 5];

	reg_dst_ij_10 = dst_ij[1 * Katz_Block_Size + 0];
	reg_dst_ij_11 = dst_ij[1 * Katz_Block_Size + 1];
	reg_dst_ij_12 = dst_ij[1 * Katz_Block_Size + 2];
	reg_dst_ij_13 = dst_ij[1 * Katz_Block_Size + 3];
	reg_dst_ij_14 = dst_ij[1 * Katz_Block_Size + 4];
	reg_dst_ij_15 = dst_ij[1 * Katz_Block_Size + 5];

	reg_dst_ij_20 = dst_ij[2 * Katz_Block_Size + 0];
	reg_dst_ij_21 = dst_ij[2 * Katz_Block_Size + 1];
	reg_dst_ij_22 = dst_ij[2 * Katz_Block_Size + 2];
	reg_dst_ij_23 = dst_ij[2 * Katz_Block_Size + 3];
	reg_dst_ij_24 = dst_ij[2 * Katz_Block_Size + 4];
	reg_dst_ij_25 = dst_ij[2 * Katz_Block_Size + 5];

	reg_dst_ij_30 = dst_ij[3 * Katz_Block_Size + 0];
	reg_dst_ij_31 = dst_ij[3 * Katz_Block_Size + 1];
	reg_dst_ij_32 = dst_ij[3 * Katz_Block_Size + 2];
	reg_dst_ij_33 = dst_ij[3 * Katz_Block_Size + 3];
	reg_dst_ij_34 = dst_ij[3 * Katz_Block_Size + 4];
	reg_dst_ij_35 = dst_ij[3 * Katz_Block_Size + 5];

	reg_dst_ij_40 = dst_ij[4 * Katz_Block_Size + 0];
	reg_dst_ij_41 = dst_ij[4 * Katz_Block_Size + 1];
	reg_dst_ij_42 = dst_ij[4 * Katz_Block_Size + 2];
	reg_dst_ij_43 = dst_ij[4 * Katz_Block_Size + 3];
	reg_dst_ij_44 = dst_ij[4 * Katz_Block_Size + 4];
	reg_dst_ij_45 = dst_ij[4 * Katz_Block_Size + 5];

	reg_dst_ij_50 = dst_ij[5 * Katz_Block_Size + 0];
	reg_dst_ij_51 = dst_ij[5 * Katz_Block_Size + 1];
	reg_dst_ij_52 = dst_ij[5 * Katz_Block_Size + 2];
	reg_dst_ij_53 = dst_ij[5 * Katz_Block_Size + 3];
	reg_dst_ij_54 = dst_ij[5 * Katz_Block_Size + 4];
	reg_dst_ij_55 = dst_ij[5 * Katz_Block_Size + 5];

	k_block *= Katz_Block_Size;
	for (int k = 0; k < blockDim.x * SIZE_COL_PER_THREAD_KATZ; k++)
	{
		reg_src_ik_0_ = src_ik[0 * Katz_Block_Size + k];
		reg_src_ik_1_ = src_ik[1 * Katz_Block_Size + k];
		reg_src_ik_2_ = src_ik[2 * Katz_Block_Size + k];
		reg_src_ik_3_ = src_ik[3 * Katz_Block_Size + k];
		reg_src_ik_4_ = src_ik[4 * Katz_Block_Size + k];
		reg_src_ik_5_ = src_ik[5 * Katz_Block_Size + k];

		reg_src_kj__0 = src_kj[k * Katz_Block_Size + 0];
		reg_src_kj__1 = src_kj[k * Katz_Block_Size + 1];
		reg_src_kj__2 = src_kj[k * Katz_Block_Size + 2];
		reg_src_kj__3 = src_kj[k * Katz_Block_Size + 3];
		reg_src_kj__4 = src_kj[k * Katz_Block_Size + 4];
		reg_src_kj__5 = src_kj[k * Katz_Block_Size + 5];
		
		reg_dst_ij_00 = fminf(reg_dst_ij_00, reg_src_ik_0_ + reg_src_kj__0);
		reg_dst_ij_01 = fminf(reg_dst_ij_01, reg_src_ik_0_ + reg_src_kj__1);
		reg_dst_ij_02 = fminf(reg_dst_ij_02, reg_src_ik_0_ + reg_src_kj__2);
		reg_dst_ij_03 = fminf(reg_dst_ij_03, reg_src_ik_0_ + reg_src_kj__3);
		reg_dst_ij_04 = fminf(reg_dst_ij_04, reg_src_ik_0_ + reg_src_kj__4);
		reg_dst_ij_05 = fminf(reg_dst_ij_05, reg_src_ik_0_ + reg_src_kj__5);

		reg_dst_ij_10 = fminf(reg_dst_ij_10, reg_src_ik_1_ + reg_src_kj__0);
		reg_dst_ij_11 = fminf(reg_dst_ij_11, reg_src_ik_1_ + reg_src_kj__1);
		reg_dst_ij_12 = fminf(reg_dst_ij_12, reg_src_ik_1_ + reg_src_kj__2);
		reg_dst_ij_13 = fminf(reg_dst_ij_13, reg_src_ik_1_ + reg_src_kj__3);
		reg_dst_ij_14 = fminf(reg_dst_ij_14, reg_src_ik_1_ + reg_src_kj__4);
		reg_dst_ij_15 = fminf(reg_dst_ij_15, reg_src_ik_1_ + reg_src_kj__5);

		reg_dst_ij_20 = fminf(reg_dst_ij_20, reg_src_ik_2_ + reg_src_kj__0);
		reg_dst_ij_21 = fminf(reg_dst_ij_21, reg_src_ik_2_ + reg_src_kj__1);
		reg_dst_ij_22 = fminf(reg_dst_ij_22, reg_src_ik_2_ + reg_src_kj__2);
		reg_dst_ij_23 = fminf(reg_dst_ij_23, reg_src_ik_2_ + reg_src_kj__3);
		reg_dst_ij_24 = fminf(reg_dst_ij_24, reg_src_ik_2_ + reg_src_kj__4);
		reg_dst_ij_25 = fminf(reg_dst_ij_25, reg_src_ik_2_ + reg_src_kj__5);

		reg_dst_ij_30 = fminf(reg_dst_ij_30, reg_src_ik_3_ + reg_src_kj__0);
		reg_dst_ij_31 = fminf(reg_dst_ij_31, reg_src_ik_3_ + reg_src_kj__1);
		reg_dst_ij_32 = fminf(reg_dst_ij_32, reg_src_ik_3_ + reg_src_kj__2);
		reg_dst_ij_33 = fminf(reg_dst_ij_33, reg_src_ik_3_ + reg_src_kj__3);
		reg_dst_ij_34 = fminf(reg_dst_ij_34, reg_src_ik_3_ + reg_src_kj__4);
		reg_dst_ij_35 = fminf(reg_dst_ij_35, reg_src_ik_3_ + reg_src_kj__5);

		reg_dst_ij_40 = fminf(reg_dst_ij_40, reg_src_ik_4_ + reg_src_kj__0);
		reg_dst_ij_41 = fminf(reg_dst_ij_41, reg_src_ik_4_ + reg_src_kj__1);
		reg_dst_ij_42 = fminf(reg_dst_ij_42, reg_src_ik_4_ + reg_src_kj__2);
		reg_dst_ij_43 = fminf(reg_dst_ij_43, reg_src_ik_4_ + reg_src_kj__3);
		reg_dst_ij_44 = fminf(reg_dst_ij_44, reg_src_ik_4_ + reg_src_kj__4);
		reg_dst_ij_45 = fminf(reg_dst_ij_45, reg_src_ik_4_ + reg_src_kj__5);

		reg_dst_ij_50 = fminf(reg_dst_ij_50, reg_src_ik_5_ + reg_src_kj__0);
		reg_dst_ij_51 = fminf(reg_dst_ij_51, reg_src_ik_5_ + reg_src_kj__1);
		reg_dst_ij_52 = fminf(reg_dst_ij_52, reg_src_ik_5_ + reg_src_kj__2);
		reg_dst_ij_53 = fminf(reg_dst_ij_53, reg_src_ik_5_ + reg_src_kj__3);
		reg_dst_ij_54 = fminf(reg_dst_ij_54, reg_src_ik_5_ + reg_src_kj__4);
		reg_dst_ij_55 = fminf(reg_dst_ij_55, reg_src_ik_5_ + reg_src_kj__5);

		if (blockIdx.x * blockDim.x + threadIdx.x == (k_block + k + 1) / SIZE_COL_PER_THREAD_KATZ || blockIdx.y * blockDim.y + threadIdx.y == (k_block + k + 1) / SIZE_ROW_PER_THREAD_KATZ)
		//if (0)
		{
			dst_ij[0 * Katz_Block_Size + 0] = reg_dst_ij_00;
			dst_ij[0 * Katz_Block_Size + 1] = reg_dst_ij_01;
			dst_ij[0 * Katz_Block_Size + 2] = reg_dst_ij_02;
			dst_ij[0 * Katz_Block_Size + 3] = reg_dst_ij_03;
			dst_ij[0 * Katz_Block_Size + 4] = reg_dst_ij_04;
			dst_ij[0 * Katz_Block_Size + 5] = reg_dst_ij_05;

			dst_ij[1 * Katz_Block_Size + 0] = reg_dst_ij_10;
			dst_ij[1 * Katz_Block_Size + 1] = reg_dst_ij_11;
			dst_ij[1 * Katz_Block_Size + 2] = reg_dst_ij_12;
			dst_ij[1 * Katz_Block_Size + 3] = reg_dst_ij_13;
			dst_ij[1 * Katz_Block_Size + 4] = reg_dst_ij_14;
			dst_ij[1 * Katz_Block_Size + 5] = reg_dst_ij_15;

			dst_ij[2 * Katz_Block_Size + 0] = reg_dst_ij_20;
			dst_ij[2 * Katz_Block_Size + 1] = reg_dst_ij_21;
			dst_ij[2 * Katz_Block_Size + 2] = reg_dst_ij_22;
			dst_ij[2 * Katz_Block_Size + 3] = reg_dst_ij_23;
			dst_ij[2 * Katz_Block_Size + 4] = reg_dst_ij_24;
			dst_ij[2 * Katz_Block_Size + 5] = reg_dst_ij_25;

			dst_ij[3 * Katz_Block_Size + 0] = reg_dst_ij_30;
			dst_ij[3 * Katz_Block_Size + 1] = reg_dst_ij_31;
			dst_ij[3 * Katz_Block_Size + 2] = reg_dst_ij_32;
			dst_ij[3 * Katz_Block_Size + 3] = reg_dst_ij_33;
			dst_ij[3 * Katz_Block_Size + 4] = reg_dst_ij_34;
			dst_ij[3 * Katz_Block_Size + 5] = reg_dst_ij_35;

			dst_ij[4 * Katz_Block_Size + 0] = reg_dst_ij_40;
			dst_ij[4 * Katz_Block_Size + 1] = reg_dst_ij_41;
			dst_ij[4 * Katz_Block_Size + 2] = reg_dst_ij_42;
			dst_ij[4 * Katz_Block_Size + 3] = reg_dst_ij_43;
			dst_ij[4 * Katz_Block_Size + 4] = reg_dst_ij_44;
			dst_ij[4 * Katz_Block_Size + 5] = reg_dst_ij_45;

			dst_ij[5 * Katz_Block_Size + 0] = reg_dst_ij_50;
			dst_ij[5 * Katz_Block_Size + 1] = reg_dst_ij_51;
			dst_ij[5 * Katz_Block_Size + 2] = reg_dst_ij_52;
			dst_ij[5 * Katz_Block_Size + 3] = reg_dst_ij_53;
			dst_ij[5 * Katz_Block_Size + 4] = reg_dst_ij_54;
			dst_ij[5 * Katz_Block_Size + 5] = reg_dst_ij_55;
		}
		__syncthreads();	
	}

	dst_ij[0 * Katz_Block_Size + 0] = reg_dst_ij_00;
	dst_ij[0 * Katz_Block_Size + 1] = reg_dst_ij_01;
	dst_ij[0 * Katz_Block_Size + 2] = reg_dst_ij_02;
	dst_ij[0 * Katz_Block_Size + 3] = reg_dst_ij_03;
	dst_ij[0 * Katz_Block_Size + 4] = reg_dst_ij_04;
	dst_ij[0 * Katz_Block_Size + 5] = reg_dst_ij_05;

	dst_ij[1 * Katz_Block_Size + 0] = reg_dst_ij_10;
	dst_ij[1 * Katz_Block_Size + 1] = reg_dst_ij_11;
	dst_ij[1 * Katz_Block_Size + 2] = reg_dst_ij_12;
	dst_ij[1 * Katz_Block_Size + 3] = reg_dst_ij_13;
	dst_ij[1 * Katz_Block_Size + 4] = reg_dst_ij_14;
	dst_ij[1 * Katz_Block_Size + 5] = reg_dst_ij_15;

	dst_ij[2 * Katz_Block_Size + 0] = reg_dst_ij_20;
	dst_ij[2 * Katz_Block_Size + 1] = reg_dst_ij_21;
	dst_ij[2 * Katz_Block_Size + 2] = reg_dst_ij_22;
	dst_ij[2 * Katz_Block_Size + 3] = reg_dst_ij_23;
	dst_ij[2 * Katz_Block_Size + 4] = reg_dst_ij_24;
	dst_ij[2 * Katz_Block_Size + 5] = reg_dst_ij_25;

	dst_ij[3 * Katz_Block_Size + 0] = reg_dst_ij_30;
	dst_ij[3 * Katz_Block_Size + 1] = reg_dst_ij_31;
	dst_ij[3 * Katz_Block_Size + 2] = reg_dst_ij_32;
	dst_ij[3 * Katz_Block_Size + 3] = reg_dst_ij_33;
	dst_ij[3 * Katz_Block_Size + 4] = reg_dst_ij_34;
	dst_ij[3 * Katz_Block_Size + 5] = reg_dst_ij_35;

	dst_ij[4 * Katz_Block_Size + 0] = reg_dst_ij_40;
	dst_ij[4 * Katz_Block_Size + 1] = reg_dst_ij_41;
	dst_ij[4 * Katz_Block_Size + 2] = reg_dst_ij_42;
	dst_ij[4 * Katz_Block_Size + 3] = reg_dst_ij_43;
	dst_ij[4 * Katz_Block_Size + 4] = reg_dst_ij_44;
	dst_ij[4 * Katz_Block_Size + 5] = reg_dst_ij_45;

	dst_ij[5 * Katz_Block_Size + 0] = reg_dst_ij_50;
	dst_ij[5 * Katz_Block_Size + 1] = reg_dst_ij_51;
	dst_ij[5 * Katz_Block_Size + 2] = reg_dst_ij_52;
	dst_ij[5 * Katz_Block_Size + 3] = reg_dst_ij_53;
	dst_ij[5 * Katz_Block_Size + 4] = reg_dst_ij_54;
	dst_ij[5 * Katz_Block_Size + 5] = reg_dst_ij_55;
}