
#include <hip/hip_runtime.h>
__global__ void Transpose_ker(float * dst, float * src, int size)
{
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size && j < size)
		dst[i * size + j] = src[j * size + i];
}

void cuTranspose(float * dst, float * src, int size)
{
	size = (size + 16 - 1) / 16 * 16;
	dim3 dimBlock(16, 16); 
	dim3 dimGrid(size / 16, size / 16);
	Transpose_ker<<<dimGrid, dimBlock>>>(dst, src, size);
}