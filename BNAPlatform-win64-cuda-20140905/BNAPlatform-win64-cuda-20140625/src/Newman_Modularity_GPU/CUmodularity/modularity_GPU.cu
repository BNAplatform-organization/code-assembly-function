#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <iomanip>
#include <memory.h>
#include <fstream>
#include <cstring>
#include "dirent.h" 
#include ""  //ͬ�������Ĳ����߲�������
#include "hip/device_functions.h"
#include "modularity_GPU.cuh"
#include <cmath>
#include <time.h> 
#include "hipblas.h"
#include "hipsparse.h"
using namespace std;

void Maslov(int * R_dst, int * C_dst, int * R_src, int * C_src, int Rlength, int Clength);

#define RANDOM_V0

extern long long N, Ntemp;
extern double * v, *vv; 
extern double * v0, * verr; 
extern	double * sumBG;
extern long long seed;

const int MAX_ITER=10000 ;			// The maximum iteration times in the power method
const int ITERNUMBER=500;
const double BETA_Adjust = 0;		// An optional parameter for quicker convergence. Its effect is uncertain
const double Epsilon = 0.000001;	// If |x - x0| < Epsilon, quit iteraion 
const double LAMBDA = 0.01;		// if labmda > LAMBDA, initiate the division
const int MIN_GROUP = 1;			// The minimum nodes of an allowed module 
extern ofstream fout;	

const int threadnumx = 16;
const int threadnumy = 16;
const int  threadnum = 256;
const int blocknum    = 48;
int* d_AD_init;
int* d_AD;
int* d_r; 
int* d_c;
int* d_orir;
double* d_u;
double* d_u0;
double* d_uu;
double * d_sumBG;
double * d_norm;
//bool* d_G;
double * temp_result;
double * d_vector;
double * d_vector1;
double * d_vector2;
double *d_k;
double *d_orik;

void Partition(int * R, int * C, int * Result);
bool Sub_Partition(int * OriR, int * OriC, int * R, int * C, int M, long long innerM, int * Result, int * Max_Result,int * AD);
double Lead_Vector(int * OriR, int * R, int * C, int M, double * sumBG1, double beta, int *AD, double *v, double *vv);

__global__ void init_AD(long N, int *AD)
{
	int tid = blockDim.x*blockIdx.x+threadIdx.x;  //����һά�ģ�
	for (int i = tid; i<N; i+=blockDim.x*gridDim.x) // ���ż� ������
		AD[i] = i;
}

__global__ void cal_k_kernel(long N, int *AD,int *d_OriR, double *d_K )
{		
	int offset;
	const int blockid   = blockIdx.x;
	const int threadid = threadIdx.x;
	
	for(offset=(blockid/2)*threadnum*2+threadid*2+blockid%2; offset<N; offset+=blockDim.x*gridDim.x )
		d_K[offset]= (double)(d_OriR[AD[offset]+1]-d_OriR[AD[offset]]);
}

__global__ void sum_kj_kernel(long N, double *d_K ,double *sum_kj)
{	__shared__ int sum[threadnum];
	int temp=0;
	int offset;
   const int threadid =blockIdx.x*blockDim.x + threadIdx.x;
	
	//for(offset=(blockid/2)*threadnum*2+threadid*2+blockid%2; offset<N; offset+=blockDim.x*gridDim.x )
		// temp+= dG[offset]*(d_R[offset+1]-d_R[offset]);
	//for (offset=blockid*threadnum; offset+threadid<N; offset+=blockDim.x*gridDim.x ) 
	 	//  temp +=dG[offset+threadid]*(d_R[offset + threadid+1]-d_R[offset + threadid]);	 		
	
	for(offset=threadid; offset<N; offset+=blockDim.x*gridDim.x)
	{
		temp+=(int)d_K[offset] ;
	}
	sum[threadIdx.x]=temp;
	syncthreads();
	
	for(offset=1;offset+threadIdx.x<threadnum;offset*=2){
			if (threadIdx.x%(2*offset)==0)  sum[threadIdx.x]+=sum[threadIdx.x+offset] ;
			syncthreads();
	}
	if(threadIdx.x==0)
	  sum_kj[blockIdx.x]=(double) sum[threadIdx.x];
}

__global__ void sum_kernel(int size, double *data, double scale)		   //��һЩǰ������������νṹ���ÿ��block�Ľ����͡�
{	
	__shared__ double sum[blocknum];
	int offset;
	sum[threadIdx.x]=data[threadIdx.x];
	syncthreads();
	
	for(offset=1;offset+threadIdx.x<blocknum; offset*=2){
		if (threadIdx.x%(2*offset)==0)  sum[threadIdx.x]+=sum[threadIdx.x+offset] ;
		syncthreads();
	}
	if(threadIdx.x==0)
	    data[0]=sum[0]*scale; 
}
               //������������  �������㶮������Ӧ�����Ĳ�������˼�������滻
__global__ void spmv_one_thread(long N, long M, double *result, int *R, int *C, double *vv, double *dk, double vk, double *d_sum, double beta, double *v0)	   //����Ai*v0,   ÿ16��threads����һ��
{
	__shared__ int R_shared[threadnum+1];

	double temp1=0;
	int offset;  
	
	for (offset=blockIdx.x*threadnum+threadIdx.x;offset<N; offset+=gridDim.x*blockDim.x)
	{
		R_shared[threadIdx.x+1] = R[offset+1];
		if(threadIdx.x==0) R_shared[threadIdx.x]=R[offset] ;
		syncthreads();	 
		temp1 =0 ;
		for(int i=R_shared[threadIdx.x]; i<R_shared[threadIdx.x+1];i++)
		{		
			temp1+=vv[C[i]];			
		}		
		temp1-=vk/(2*M)*dk[offset]+(d_sum[offset]-beta)*v0[offset];
		result[offset]=temp1 ;
			//syncthreads();
	}		
}
               //�������
 __global__ void vvplus (long N, double *result, double *v0, double alpha, double *v1, double beta)    //���������ӷ�
 {
	 const int blockid   = blockIdx.x;
	 const int threadid  = threadIdx.x;
	 int offset;  	 
	 for(offset=threadid+blockid*threadnum; offset<N; offset+=threadnum*gridDim.x)
		 result[offset]=(alpha*v0[offset]+beta*v1[offset]);
 }
 
 __global__ void sumBG_kernel (double *sumBG, double *d_orik, double *d_k, int *AD,long N, double innerMd2M)    //���������ӷ�
 {
	 //sumBG[i] = R[i+1] - R[i] - (OriR[AD[i]+1] - OriR[AD[i]]) * (double)innerM / 2 / M;
	 const int blockid   = blockIdx.x;
	 const int threadid  = threadIdx.x;
	 int offset;  	 
	 for(offset=threadid+blockid*threadnum; offset<N; offset+=threadnum*gridDim.x)
	 {
		sumBG[offset] = d_k[offset] - (d_orik[AD[offset]]) * innerMd2M;
	 }
 }
 
 __global__ void cal_VV_kernel ( long Ntemp, long N, int *AD, double *vv, double *v)
 {
	 const int blockid   = blockIdx.x;
	 const int threadid  = threadIdx.x;
	 int offset;
	 for(offset=threadid+blockid*threadnum; offset<Ntemp; offset+=threadnum*gridDim.x)
		 vv[AD[offset]] = v[offset];
 }

 __global__ void calc_vector (long N, double *result, double *dk, double vk2m, double *d_sum, double beta, double *v0)	   //���� ( v_k/2/M * (R[i+1] - R[i])+(sumBG1[i] - beta) * v0[i])��
 {		
	 const int blockid   = blockIdx.x;
	 const int threadid = threadIdx.x;
	 int offset;
	 double temp=0;
	 
	 for(offset=threadnum*blockid+threadid; offset<N; offset+=threadnum*gridDim.x)
	 {	
		  temp=vk2m*dk[offset]+(d_sum[offset]-beta)*v0[offset];
		  result[offset]=temp;
	 }
  }

 /*__global__ void  Norm2_ph1(long N, double *norm,  double *v, bool *dG)	 //�������Ķ����������sum_kernel �õ����ս��
 {
	 __shared__  double temp[threadnum];
	 const int blockid   = blockIdx.x;
	 const int threadid = threadIdx.x;
	 double temp1=0;
	 int offset;

	 for(offset=blockid*threadnum+threadid; offset<N; offset+=threadnum*gridDim.x)
		 temp1+=dG[offset]? v[offset]*v[offset] : 0;
	 temp[threadid]=temp1;
	 syncthreads();

	 for(offset=1;offset+threadid<threadnum;offset*=2){
		 if (threadid%(2*offset)==0)  temp[threadid]+=temp[threadid+offset] ;
		 syncthreads();
	 }
	 if (threadid==0)
		 norm[blockid] = temp[threadid];  	 
 }
 __global__ void Norm2_ph2(long N,  double norm, double *v)						  //��������һ������Ϊ����������ÿ��Ԫ�س���1�����ֲ��䡣
 {	 	 
	 for (int offset = threadIdx.x+blockIdx.x*threadnum; offset<N ; offset+= threadnum*gridDim.x  )
		 v[offset]/=(norm? (norm) : 1);  
 } */







/* 
This function returns the norm of the input vector x[G].
G is the logic subscriber and N is the matrix dimension.
*/


double Lead_Vector_GPU(int * OriR, int *R, int *C,  int M, double beta, int *AD)
{
	long long i = 0, j = 0;
	/*double *k = new double [Ntemp];
	for (int p=0; p<Ntemp; p++)
	{
		k[p] = p;
	}*/
	// Initialize v. Two methods are optional. Define RANDOM_V0 if you want to use random starting vector
#ifdef RANDOM_V0
	//srand(time(0));
	srand(seed);
	for (i = 0; i < Ntemp; i++){
		v[i] = AD[i];
		//if(i<20) cout<<v[i]<<endl;
	}	 
	for (i = 0; i < N; i++){
		vv[i] = 0;
		//if(i<20) cout<<v[i]<<endl;
	}
#else
	for (i = 0; i < N && !G[i]; i++)
#endif
	hipError_t cudaStat;
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	stat = hipblasCreate(&handle) ;
	checkCudaErrors( hipMemset (d_u, 0, sizeof(double) * (Ntemp)));
	checkCudaErrors( hipMemcpy( d_u, v, sizeof(double) * Ntemp , hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy( d_uu, vv, sizeof(double) * N , hipMemcpyHostToDevice) );
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;

	double err1 = 1, err2 = 1;
	int ITER = 0;
	double vNorm = 0;
	double temp2= -1;
	double temp1=0;
	//double *norm_v= new double ;
	//double *check_du= new double [N];
    double v_k;
	
	//int blocknum_spmv = N*HALF_WARP/threadnum+1;

	//spmv_kernel<<<blocknum_spmv, threadnum>>>((long) N, d_r, d_c, d_G, d_u, d_u0);
	dim3 blocknum_spmv ( Ntemp/threadnumy+(Ntemp%threadnumy?1:0) );
	dim3 threadn(threadnumx,threadnumy);
	//cal_k_kernel<<<6,threadnum>>>((long) Ntemp, d_AD, d_orir, d_k);

	while (err1 > Epsilon &&  err2 > Epsilon && ITER < MAX_ITER)
	{	  		
		
	   hipblasDcopy(handle, (int) Ntemp, d_u, 1 ,d_u0, 1 );
	   //������Ҫ����vv��
	   cal_VV_kernel<<<blocknum,threadnum>>>((long) Ntemp, (long) N, d_AD, d_uu, d_u);
	   
	   hipblasDdot (handle, Ntemp, d_u0, 1, d_k, 1, &v_k);
	   //calc_vector<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_k, v_k/(2*M), d_sumBG, beta, d_u0);
	  /* checkCudaErrors( hipMemcpy( vvector, d_vector, sizeof(double) * (Ntemp), hipMemcpyDeviceToHost) );
	   for (int ii = 0;ii < Ntemp;ii++)
	   {
		   cout<<"vector["<<ii<<"] = "<<vvector[ii]<<endl;
	   }*/
  		spmv_one_thread<<<blocknum , threadnum>>>((long)Ntemp, (long) M, d_u, d_r, d_c, d_uu, d_k, v_k, d_sumBG, beta, d_u0) ;
		/*checkCudaErrors( hipMemcpy( v, d_u, sizeof(double) * Ntemp , hipMemcpyDeviceToHost) ); 
	  for (int ii = 0;ii < Ntemp;ii++)
	   {
		   cout<<"v["<<ii<<"] = "<<v[ii]<<endl;
	   }*/
		//hipblasDaxpy(handle, (int) Ntemp, &temp2, d_vector, 1, d_u, 1);
		//checkCudaErrors( hipMemcpy( v, d_u, sizeof(double) * (Ntemp), hipMemcpyDeviceToHost) );
	   /*for(int ii=0;ii<Ntemp;ii++)
	   {
			if (v[ii]!=0)
				cout<<"v["<<ii<<"] = "<<v[ii]<<endl;
	   }*/
	   
	    hipblasDnrm2(handle, Ntemp, d_u, 1, &vNorm);
		temp1=1/vNorm;
		hipblasDscal (handle, (int) Ntemp, &temp1, d_u, 1);   //Normalize v, v[i] = v[i]/vNorm
	
		vvplus<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_u, 1.0, d_u0, -1.0);
	    hipblasDnrm2(handle, Ntemp, d_vector, 1, &err1);
		vvplus<<<blocknum, threadnum>>>((long) Ntemp, d_vector, d_u, 1.0, d_u0, 1.0);
		hipblasDnrm2(handle, Ntemp, d_vector, 1, &err2);
				 
		ITER++;
	}	 
	//system("pause");
	cout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<min(err1, err2)<<'\t';
	fout<<"Iterations:\t"<<ITER<<'\t'<<"residual:\t"<<min(err1, err2)<<'\t';
	
	checkCudaErrors( hipMemcpy( v, d_u, sizeof(double) * Ntemp , hipMemcpyDeviceToHost) ); 
	checkCudaErrors( hipMemcpy(v0,d_u0, sizeof(double) * Ntemp,  hipMemcpyDeviceToHost) );
	hipblasDestroy(handle);
	long long max_index = 0;
	for (i = 0; i < Ntemp; i++)
		if (fabs(v[i]) > fabs(v[max_index]))
			max_index = i;
	return (v[max_index] * v0[max_index] > 0) ? vNorm: -vNorm;
}



bool Sub_Partition_GPU(int * OriR, int * OriC, int * R, int * C, int M, int innerM, int * Result, int * Max_Result,int * AD)
{
	//double *v = new double [Ntemp];
	//double *vv = new double [Ntemp];
	long long i = 0, j = 0;
	long long temp1,temp2;
    double *sumBG1= new double [Ntemp];
	hipblasStatus_t stat;
	hipblasHandle_t handle;
	stat = hipblasCreate(&handle) ;
	if (stat != HIPBLAS_STATUS_SUCCESS)
		return stat;
	//dim3 blocknum_sumBG(  Ntemp/threadnumy+(Ntemp%threadnumy?1:0) ) ;
	//dim3 threadn(threadnumx,threadnumy);
	checkCudaErrors( hipMemset (d_sumBG, 0, sizeof(double) * (Ntemp)));
	//sum_kj_kernel <<<blocknum, threadnum>>> ((long) N,  d_k, temp_result);
	//sum_kernel<<<1,blocknum>>>(blocknum, temp_result, 1); 
	/*for (i = 0; i < Ntemp; i++)
	{
		sumBG[i] = 0;
		sumBG[i] = R[i+1] - R[i] - (OriR[AD[i]+1] - OriR[AD[i]]) * (double)innerM / 2 / M;
	}*/
	cal_k_kernel<<<blocknum,threadnum>>>((long) Ntemp, d_AD_init, d_r, d_sumBG);
	cal_k_kernel<<<blocknum,threadnum>>>((long) Ntemp, d_AD, d_orir, d_k);
	double innerMd2M = 0.0 - (double) innerM/2/M;
	hipblasDaxpy(handle, (int) Ntemp, &innerMd2M, d_k, 1, d_sumBG, 1);
	//sumBG_kernel<<< blocknum, threadnum >>>(d_sumBG, d_orik, d_k, d_AD, (long) Ntemp, (double)innerM/2/M);
	//checkCudaErrors( hipMemcpy( sumBG1, d_sumBG, sizeof(double)*(Ntemp), hipMemcpyDeviceToHost)) ;
	//checkCudaErrors(hipMemcpy( d_sumBG, sumBG, sizeof(double)*(Ntemp), hipMemcpyHostToDevice)) ;
	//checkCudaErrors( hipMemcpy( sumBG, d_sumBG, sizeof(double)*(Ntemp), hipMemcpyDeviceToHost)) ;
	//cout<<"innerM = "<<innerM;
	/*for(int ii=0;ii<Ntemp;ii++)
	{
		if ((sumBG1[ii]-sumBG[ii])>Epsilon || sumBG1[ii]-sumBG[ii]<-Epsilon)
			cout<<"sumBG["<<ii<<"] = "<<sumBG[ii]<<" sumBG1["<<ii<<"] = "<<sumBG1[ii]<<endl;
    }*/
	double lambda = 0;
	lambda = Lead_Vector_GPU(OriR, R, C, M, 0, AD);
	lambda -= BETA_Adjust;
	// If lambda < 0, calucate the leading eigenvalue for  B - lambda * I
	if (lambda < (-1.0)*LAMBDA)
		//lambda += Lead_Vector_GPU(  M,  G, -lambda);
		lambda += Lead_Vector_GPU(OriR, R, C, M, -lambda, AD);
		//lambda = Lead_Vector(R, C, M, sumBG, G, -lambda);
	cout<<"Eigen Value: "<<lambda<<'\t';
	fout<<"Eigen Value: "<<lambda<<'\t';

	// Decide whether this round of partition is successful 
	long long subN = 0, subP = 0;
	for (i = 0; i < Ntemp; i++)
	{
		subP += (v[i] > 0);
		subN += (v[i] <= 0);
	}
	bool Issub = (lambda > LAMBDA && subP > MIN_GROUP && subN > MIN_GROUP);

	cout<<"Divide?: "<<Issub<<"\t\n";
	fout<<"Divide?: "<<Issub<<"\t\n";
	// If not divided, return; otherwise update Result and Max_Result
	if (!Issub)
		return 0;
	for (i = 0; i < Ntemp; i++)
		Result[AD[i]] = *Max_Result + 1 + (v[i] * (subP - subN + 0.5) <= 0);
	// notice: this is wrong  Result[i] = *Max_Result + 1 + (v[i] * (subP - subN) >= 0);
	(*Max_Result) += 2;
	//delete []v;
	//delete []vv;
	return Issub;
}


/* 
This function does the partition, no return value.
R and C represent the adjacency matrix in CSR format.
Result stores the partition results.
*/
void Partition_GPU(int * R, int * C, int * Result)
{   
	int devID;
	hipDeviceProp_t deviceProps;
	devID = findCudaDevice();
	// get number of SMs on this GPU
	checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
	//printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);
   

	
	Setup(0);
	Start(0);
	int M = (R[N] - R[0]) / 2;			// The total Number connection in the network
	int Round = 0;						// The iteration round
	int Max_Result = 0;					// Maximum index of modules
	int Module_Num = 1;					// Used for adjust module index.
	bool Issub;							// Return by function Sub_Partition()
	bool * G = new bool [N];			// G[i] = 1 if node i is involved in this round of partition
	memset(Result, 0, sizeof(int) * N);
	int * Adjust_Result = new int [N];	// Map the results to consecutive intergers starting from 1 
	int Index = 1;						// Used in the adjusted results, starting from 1, increase by 1 at each successful division
	int * NewRow = new int [N+1];
	int * NewCol = new int [R[N]];
	int * Index_Result = new int [N];  // Used for matching the order number for every round partition
	long long i = 0, j = 0;
	int NumG = 0;						//
	long long innerM = 2*M;
	int Newtemp1 = 0;
	int	Newtemp2 = 0;
	while (Round <= Max_Result)	   //???????
	{
		innerM =2*M;
		NumG = 0;
		for (int i = 0; i < N; i++)
		{
			G[i] = (Result[i] == Round); 
			if (G[i])
			{
				Index_Result[NumG] = i;
				NumG ++;				// G[i] = 1 if node i is involved in this round
				continue;
			}
			innerM -= R[i+1] - R[i];
		}
		//Select all the involved node to form new row and col
		Newtemp1 = 0;
		Newtemp2 = 0;
		NewRow[Newtemp2] = Newtemp1;
		Newtemp2++;
		for(int i = 0;i < N; i++)
		{
			if(!G[i])
				continue;
			for (int j = R[i];j < R[i+1];j++)
			{
				if(!G[C[j]])
					continue;
				NewCol[Newtemp1] = C[j];
				Newtemp1++;
			}
			NewRow[Newtemp2] = Newtemp1;
			Newtemp2++;
		}
		Ntemp = Newtemp2 - 1;
		//main part of the partition
		checkCudaErrors( hipMemcpy( d_r, NewRow, sizeof(int) * (N + 1), hipMemcpyHostToDevice)) ;
		checkCudaErrors( hipMemcpy( d_c, NewCol, sizeof(int) * (2*M), hipMemcpyHostToDevice)) ;
		checkCudaErrors( hipMemcpy( d_AD, Index_Result, sizeof(int) * (N), hipMemcpyHostToDevice)) ;
		if (NumG)						 
		{
			cout<<"\nRound:\t"<<Round<<'\t';
			cout<<"number of nodes:\t"<<NumG<<'\t';
			fout<<"\nRound:\t"<<Round<<'\t';
			fout.flush();
			fout<<"number of nodes:\t"<<NumG<<'\t';	
			Setup(1);
			Start(1);
			if (NumG>25000)
				Issub = Sub_Partition_GPU(R,C,NewRow, NewCol, M, innerM, Result, &Max_Result, Index_Result);
			else
				Issub = Sub_Partition(R,C,NewRow, NewCol, M, innerM, Result, &Max_Result, Index_Result);
										// call Sub_Partition() for this round of division
			Stop(1);
			cout<<"sub_partition time:   "<<GetElapsedTime(1)<<"s"<<endl;
			fout<<"sub_partition time:   "<<GetElapsedTime(1)<<"s"<<endl;

			if (!Issub)					// If divided, record the adjusted result
				Adjust_Result[Round] = Index++;
			Module_Num += Issub;		// Update the total number modules 
		}
		Round++;
	}
	Stop(0);

	// calculate Q
	double Q = 0;
	for (i = 0; i < N; i++)
		for (j = R[i]; j < R[i+1]; j++)
			Q += 1.0 * (Result[i] == Result[C[j]]);
	for (i = 0; i < N; i++)
		for (j = 0; j < N; j++)
			Q -= 1.0 * (Result[i] == Result[j]) * (R[i+1]-R[i]) * (R[j+1]-R[j]) / 2 / M;

	Q /= 2 * M;

	cout<<"\nNumber of Modules: "<<Module_Num<<",\tQ="<<Q<<endl;
	fout<<"\nNumber of Modules: "<<Module_Num<<",\tQ="<<Q<<endl;
	cout<<"Elapsed time:   "<<GetElapsedTime(0)<<"s"<<endl;
	fout<<"Elapsed time:   "<<GetElapsedTime(0)<<"s"<<endl;

	// Adjust the results
	for (i = 0; i < N; i++)	
		Result[i] = Adjust_Result[Result[i]];
	delete []Adjust_Result;
	delete []G;
	delete []Index_Result;
	delete []NewRow;
	delete []NewCol;
	return;
}

int main(int argc, char * argv[])
{
	ofstream flog("BNA_time_log", ios::app);//ofstream��stream�����࣬���ڴ浽Ӳ�̣�ios����app��׷�ӷ�ʽ���ļ�
	clock_t total_time = clock();
	if (argc != 3) 
	{
		cerr<<"Input format: .\\Modularity.exe dir_for_csr num_of_random_networks \nFor example: .\\Modularity_CPU.exe d:\\data 10"<<endl;
		exit(1);	
	   //cerr��cout����Ҫ���־���,cout�������Ϣ�����ض���,��cerrֻ���������׼���(��ʾ��)�ϡ�
	}

	DIR *dp;
	struct dirent *dirp;
	if (NULL == (dp = opendir(argv[1])))
	{
		printf("can't open %s", argv[1]);
		exit (1);
	}                 //����Щ�ļ��������ѣ������Ȳ�����
	int FileNumber = 0;
	string filenametmp;
	while((dirp = readdir(dp)) != NULL)
	{
		filenametmp = string(dirp->d_name);

		if (filenametmp.find_last_of('.') == -1)
			continue;
		if(filenametmp.length()>4 && filenametmp.substr(filenametmp.find_last_of('.'),4).compare(".csr") == 0 && filenametmp.size() - filenametmp.find_last_of('.') - 1 == 3)
		{
			FileNumber++;
		}
	}
	cout<<FileNumber<<" files to be processed."<<endl;

	closedir(dp);
	string *filename = new string[FileNumber];
	dp = opendir(argv[1]);
	long long i = 0;
	while((dirp = readdir(dp)) != NULL)
	{
		filenametmp = string(dirp->d_name);
		if (filenametmp.find_last_of('.') == -1)
			continue;
		if(filenametmp.length()>4 && filenametmp.substr(filenametmp.find_last_of('.'),4).compare(".csr") == 0 && filenametmp.size() - filenametmp.find_last_of('.') - 1 == 3)
		{
			filename[i++] = filenametmp;
		}
	}

	int max_iso_n = 0;
	for (long long i = 0; i < FileNumber; i++)
	{
		string a = string(argv[1]).append("\\").append(filename[i]);
		cout<<"\nModular analysis for "<<a.c_str()<<" ..."<<endl;
		ifstream fin(a.c_str(), ios_base::binary);
		if (!fin.good())
		{	cout<<"Can't open\t"<<a.c_str()<<endl;	return 0;}
		// Read x.csr
		int Rlength = 0, Clength = 0;
		fin.read((char*)&Rlength, sizeof(int));
		int * R = new int [Rlength];
		fin.read((char*)R, sizeof(int) * Rlength);
		fin.read((char*)&Clength, sizeof(int));
		int * C = new int [Clength];
		fin.read((char*)C, sizeof(int) * Clength);
		fin.close();
		N = Rlength - 1;

		int isolated_n = 0;
		for (int j = 0; j < N; j++)
			if (R[j]==R[j+1])
				isolated_n++;
		if (isolated_n > max_iso_n)
			max_iso_n = isolated_n;

		// allocate buffers used in the iteration
		v = new double [N];
		v0 = new double [N];
		vv = new double [N];
		verr = new double [N];
		sumBG = new double [N];
		// allocate the result buffer and call Partition()
		int * Modu_Result = new int [N];	

		// Parse file name
		string X_modu = a.substr(0, a.find_last_of('.') + 1).append("modu");
		string X_cp_mas = a.substr(0, a.find_last_of('.')).append("_modu.txt");
		fout.open(X_cp_mas.c_str(), ios::out);	// Open the log file

		checkCudaErrors( hipMalloc( (void**) &d_AD, sizeof(int) * (N)));
		checkCudaErrors( hipMalloc( (void**) &d_AD_init, sizeof(int) * (N)));
		checkCudaErrors( hipMalloc( (void**) &d_orir, sizeof(int) * (N + 1)));
		checkCudaErrors( hipMalloc( (void**) &d_r, sizeof(int) * (N + 1)));
		checkCudaErrors( hipMalloc( (void**) &d_c, sizeof(int) * (R[N]-R[0])));
		checkCudaErrors( hipMalloc( (void**) &d_u, sizeof(double) * (N)));
		checkCudaErrors( hipMalloc( (void**) &d_uu, sizeof(double) * (N)));
		checkCudaErrors( hipMalloc( (void**) &d_u0, sizeof(double) * (N)));
		checkCudaErrors( hipMalloc( (void**) &d_sumBG, sizeof(double) * (N)));
		checkCudaErrors( hipMalloc( (void**) &d_k, sizeof(double) * (N)));
		checkCudaErrors( hipMalloc( (void**) &d_orik, sizeof(double) * (N)));
		checkCudaErrors( hipMalloc( (void**) &d_vector, sizeof(double) * N));
		
	// copy host memory to device
		checkCudaErrors( hipMemcpy( d_orir, R, sizeof(int) * (N + 1), hipMemcpyHostToDevice)) ;
		init_AD<<<blocknum,threadnum>>>((long) N, d_AD_init);
		cal_k_kernel<<<blocknum,threadnum>>>((long) N, d_AD_init, d_orir, d_orik);
		//double *k= new double [N];
		//checkCudaErrors( hipMemcpy( k, d_orik, sizeof(double) * (N), hipMemcpyDeviceToHost)) ;
		//for (int x=0; x < N; x++)
		//	if ((R[x+1]-R[x]-k[x])!=0) cout<<"k["<<x<<"] = "<<R[x+1]-R[x]<<"  ;  " <<k[x] <<endl;  


		seed=time(NULL);
        Setup(0);
		Start(0);
		//Partition(R, C, Modu_Result);
		Partition_GPU(R, C, Modu_Result);
		Stop(0);
		flog<<"Modularity\t"<<a.c_str()<<"CPU\tkernel time\t"<<GetElapsedTime(0)<<"s"<<endl;

		cout<<"Save partition results as "<<X_modu.c_str()<<endl;
		ofstream fresult;
		fresult.open(X_modu.c_str(), ios::binary|ios::out);
		fresult.write((char*)&N, sizeof(int));
		fresult.write((char*)Modu_Result, sizeof(int) * N);
		fresult.close();

		// Analysis for random networks
		int Maslov_num = atoi(argv[2]);
		cout<<"Modular analysis for random networks..."<<endl;

		int * R_dst = new int [Rlength];
		int * C_dst = new int [Clength];
		Setup(0);
		Start(0);
		for (long long l = 0; l < Maslov_num; l++)
		{
			Maslov(R_dst, C_dst, R, C, Rlength, Clength);
			checkCudaErrors( hipMemcpy( d_r, R_dst, sizeof(int) * (N + 1), hipMemcpyHostToDevice)) ;
			checkCudaErrors( hipMemcpy( d_c, C_dst, sizeof(int) * ((R_dst[N]-R_dst[0])), hipMemcpyHostToDevice)) ;
			cal_k_kernel<<<6,threadnum>>>((long) N, d_AD, d_r, d_k);
			
			//Partition(R_dst, C_dst, Modu_Result);
			Partition_GPU(R_dst, C_dst, Modu_Result);
		}
		Stop(0);
		flog<<"Modularity\tRandom"<<"CPU\t(Maslov+kernel) time\t"<<GetElapsedTime(0)<<"s"<<endl;
		// Clean up
		fout.close();
		delete []Modu_Result;
		delete []sumBG;
		delete []verr;
		delete []v0;
		delete []vv;
		delete []v;
		delete []R;
		delete []C;
		delete []R_dst;
		delete []C_dst;
		checkCudaErrors(hipFree(d_r));
		checkCudaErrors(hipFree(d_k));
		checkCudaErrors(hipFree(d_orik));
		checkCudaErrors(hipFree(d_c));
		checkCudaErrors(hipFree (d_u));
		checkCudaErrors(hipFree (d_uu));
		checkCudaErrors(hipFree (d_u0));
		checkCudaErrors(hipFree (d_sumBG));
		//checkCudaErrors(hipFree (d_G));
		checkCudaErrors(hipFree (temp_result));
		checkCudaErrors(hipFree (d_vector));
		checkCudaErrors(hipFree (d_vector1));
		checkCudaErrors(hipFree (d_vector2));	
		checkCudaErrors(hipFree (d_norm));
		checkCudaErrors(hipFree (d_AD));
		checkCudaErrors(hipFree (d_AD_init));
	}
	cout<<"==========================================================="<<endl;
	total_time = clock() - total_time;
	flog<<"Modularity\tCPU\ttotal time\t"<<1.0*total_time/1000<<"s"<<endl;
	flog<<"seed="<<seed<<endl;
	cout<<"max isolated voxel number: "<<max_iso_n;
	flog<<endl;
	flog.close();
	//system("pause");
	delete[]filename;
	return 0;
}	   
