#include "hip/hip_runtime.h"
#include "BC_GPU.cuh"

// changeable for performance 
#define VIRTUAL_WARP 16

__global__ void betweenness_node_pred_kernel(float *BC, int * r, int * edge_begin, int *edge_end, int * dist, float * sigma, float * delta, int * P, int * P_end, int numVertices, int numEdges, int offset_source) 
{
	int offset_vertices = blockIdx.x * numVertices;
	int offset_edge = blockIdx.x * numEdges;

	for (int i = threadIdx.x; i < numVertices; i += blockDim.x)
	{
		dist[offset_vertices + i] = -1;
		sigma[offset_vertices + i] = 0;
		delta[offset_vertices + i] = 0;

		P_end[offset_vertices + i] = r[i];
	}

	for (int i = threadIdx.x; i < numEdges; i += blockDim.x)
	{
		P[offset_edge + i] = 0;
	}

	int source = blockIdx.x + offset_source;
	if (source >= numVertices)
		return;
	__shared__ bool done;
	done = false;
	
	int level = 0;
	dist[offset_vertices + source] = level++;
	sigma[offset_vertices + source ] = 1; 

	while (!done)
	{
		__syncthreads(); // attention: this sync is neccessary
		done = true;
		for (int edge = threadIdx.x; edge < numEdges; edge += blockDim.x)
		{
			int current = edge_begin[edge];
			if (dist[offset_vertices + current] != level - 1)
				continue;
			
			int next = edge_end[edge];
			int read_dist = dist[offset_vertices + next];
			if (read_dist == -1)
			{
				dist[offset_vertices + next] = level;
				done = false;
			}
			if (read_dist < level && read_dist >= 0)
				continue;

			atomicAdd(sigma + offset_vertices + next, sigma[offset_vertices + current]); //atomic!

			int p = atomicAdd(P_end + offset_vertices + next, 1);
			P[offset_edge + p] = current;
			
		}
		level ++;
		__syncthreads();
	}

	for (int i = level - 1; i > 0; i--)
	{
		for (int next = threadIdx.x; next < numVertices; next += blockDim.x)
		{
			if (dist[offset_vertices + next] != i)
				continue;
			for (int j = r[next]; j < P_end[offset_vertices + next]; j += 1)
			{
				int current = P[offset_edge + j];
				atomicAdd(delta + offset_vertices + current, (double) sigma[offset_vertices + current] / sigma[offset_vertices + next]*(1 + delta[offset_vertices + next]));
			}
		}
		__syncthreads();
	}

	for (int current = threadIdx.x; current < numVertices; current += blockDim.x)
	{
		if(current != source)
			atomicAdd(BC + current, delta[offset_vertices + current]);
	}
}

void Betweenness_GPU_edge_pred(int *r, int *r_full, int *c, int numVertices, int numEdges, float *BC, int grid, int thread)
{
    int devID;
    hipDeviceProp_t deviceProps;
    devID = findCudaDevice();
    // get number of SMs on this GPU
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

	//int thread = 256;
	//int grid = 100;
    // allocate device memory
    int* d_r; 
	int* d_c;
	int* d_r_full;
	int* dist;
	float* sigma;
	float* delta;
	int* P;
	int* P_end;

    checkCudaErrors( hipMalloc( (void**) &d_r, sizeof(int) * (numVertices + 1)));
	checkCudaErrors( hipMalloc( (void**) &d_r_full, sizeof(int) * numEdges));
	checkCudaErrors( hipMalloc( (void**) &d_c, sizeof(int) * numEdges));
	checkCudaErrors( hipMalloc( (void**) &dist, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &sigma, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &delta, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &P, sizeof(int) * numEdges * grid));
	checkCudaErrors( hipMalloc( (void**) &P_end, sizeof(int) * numVertices * grid));

    // copy host memory to device
	checkCudaErrors( hipMemcpy( d_r, r, sizeof(int) * (numVertices + 1), hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy( d_c, c, sizeof(int) * numEdges, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy( d_r_full, r_full, sizeof(int) * numEdges, hipMemcpyHostToDevice) );

    // allocate device memory for result
    float* d_BC;
    checkCudaErrors( hipMalloc( (void**) &d_BC, sizeof(float) * numVertices));
	checkCudaErrors( hipMemset( d_BC, 0, sizeof(float) * numVertices));

	// execute the kernel
	clock_t kernel_time = 0;
	for (int offset_source = 0; offset_source < numVertices; offset_source += grid)
	{
		clock_t time = clock();
		betweenness_node_pred_kernel<<<grid, thread>>>(d_BC, d_r, d_r_full, d_c, dist, sigma, delta, P, P_end, numVertices, numEdges, offset_source);
		// check if kernel execution generated and error
		getLastCudaError("Kernel execution failed");
		hipDeviceSynchronize();
		time = clock() - time;

		kernel_time += time;
		cout<<offset_source<<" done. Time = "<<time<<"ms."<<endl;
	}
	cout<<"total kernel time: "<<kernel_time<<"ms."<<endl;

	// copy result from device to host
	checkCudaErrors(hipMemcpy(BC, d_BC, sizeof(float) * numVertices, hipMemcpyDeviceToHost));

    // cleanup memory
    checkCudaErrors(hipFree(d_r));
	checkCudaErrors(hipFree(d_r_full));
    checkCudaErrors(hipFree(d_c));
	checkCudaErrors(hipFree(d_BC));
	checkCudaErrors(hipFree(dist));
    checkCudaErrors(hipFree(sigma));
	checkCudaErrors(hipFree(delta));
	checkCudaErrors(hipFree(P));
	checkCudaErrors(hipFree(P_end));
    hipDeviceReset();
}
