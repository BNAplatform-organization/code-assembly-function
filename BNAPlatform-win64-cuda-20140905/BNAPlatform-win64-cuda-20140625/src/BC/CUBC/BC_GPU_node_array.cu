#include "hip/hip_runtime.h"
#include "BC_GPU.cuh"

// changeable for performance 
#define VIRTUAL_WARP 32

__global__ void betweenness_node_array_kernel(float *BC, int *r, int *c, int * dist, float * sigma, float * delta, bool * P, int numVertices, int numEdges, int offset_source) 
{
	int offset_vertices = blockIdx.x * numVertices;
	int offset_edge = blockIdx.x * numEdges;

	for (int i = threadIdx.x; i < numVertices; i += blockDim.x)
	{
		dist[offset_vertices + i] = -1;
		sigma[offset_vertices + i] = 0;
		delta[offset_vertices + i] = 0;
	}
	for (int i = threadIdx.x; i < numEdges; i += blockDim.x)
	{
		P[offset_edge + i] = false;
	}

	int edge_index = threadIdx.x % VIRTUAL_WARP;
	int vertice_index = threadIdx.x / VIRTUAL_WARP;
	int source = blockIdx.x + offset_source;
	if (source >= numVertices)
		return;
	__shared__ bool done;
	done = false;
	
	int level = 0;
	dist[offset_vertices + source] = level++;
	sigma[offset_vertices + source] = 1; 
	while (!done)
	{
		__syncthreads(); // attention: this sync is neccessary
		done = true;
		for (int current = vertice_index; current < numVertices; current += blockDim.x / VIRTUAL_WARP)
		{
			if (dist[offset_vertices + current] != level - 1)
				continue;
			for (int j = r[current] + edge_index; j < r[current + 1]; j += VIRTUAL_WARP)
			{
				int next = c[j];
				int read_dist = dist[offset_vertices + next];
				if (read_dist == -1)
				{
					dist[offset_vertices + next] = level;
					done = false;
				}
				//__syncthreads();
				if (read_dist < level && read_dist >= 0)
					continue;

				atomicAdd(sigma + offset_vertices + next, sigma[offset_vertices + current]); 
				// no atomic
				P[offset_edge + j] = true;
			}
		}
		level ++;
		__syncthreads();
	}
	for (int i = level - 2; i >= 0; i--)
	{
		// no atomic
		for (int current = threadIdx.x; current < numVertices; current += blockDim.x)
		{
			if (dist[offset_vertices + current] != i)
				continue;
			for (int j = r[current]; j < r[current + 1]; j += 1)
			{
				if (P[offset_edge + j] != true)
					continue;
				int next = c[j];
				delta[offset_vertices + current] += (double) sigma[offset_vertices + current] / sigma[offset_vertices + next]*(1 + delta[offset_vertices + next]);
			}
		}
		__syncthreads();
	}

	for (int current = threadIdx.x; current < numVertices; current += blockDim.x)
	{
		if(current != source)
			atomicAdd(BC + current, delta[offset_vertices + current]);
	}
}

void Betweenness_GPU_node_array(int *r, int *c, int numVertices, int numEdges, float *BC, int grid, int thread)
{
    int devID;
    hipDeviceProp_t deviceProps;
    devID = findCudaDevice();
    // get number of SMs on this GPU
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

	//int thread = 256;
	//int grid = 100;
    // allocate device memory
    int* d_r; 
	int* d_c;
	int* dist;
	float* sigma;
	float* delta;
	bool* P;

    checkCudaErrors( hipMalloc( (void**) &d_r, sizeof(int) * (numVertices + 1)));
	checkCudaErrors( hipMalloc( (void**) &d_c, sizeof(int) * numEdges));
	checkCudaErrors( hipMalloc( (void**) &dist, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &sigma, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &delta, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &P, sizeof(bool) * numEdges * grid));
    // copy host memory to device
	checkCudaErrors( hipMemcpy( d_r, r, sizeof(int) * (numVertices + 1), hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy( d_c, c, sizeof(int) * numEdges, hipMemcpyHostToDevice) );

    // allocate device memory for result
    float* d_BC;
    checkCudaErrors( hipMalloc( (void**) &d_BC, sizeof(float) * numVertices));
	checkCudaErrors( hipMemset( d_BC, 0, sizeof(float) * numVertices));

	clock_t kernel_time = 0;

    // execute the kernel
	for (int offset_source = 0; offset_source < numVertices; offset_source += grid)
	{
		clock_t time = clock();
		betweenness_node_array_kernel<<<grid, thread>>>(d_BC, d_r, d_c, dist, sigma, delta, P, numVertices, numEdges, offset_source);
		// check if kernel execution generated and error
		getLastCudaError("Kernel execution failed");
		hipDeviceSynchronize();
		time = clock() - time;

		cout<<offset_source<<" done. Time = "<<time<<"ms."<<endl;
		kernel_time += time;
	}

	cout<<"total kernel time: "<<kernel_time<<"ms."<<endl;

	// copy result from device to host
	checkCudaErrors(hipMemcpy(BC, d_BC, sizeof(float) * numVertices, hipMemcpyDeviceToHost));

    // cleanup memory
    checkCudaErrors(hipFree(d_r));
    checkCudaErrors(hipFree(d_c));
	checkCudaErrors(hipFree(d_BC));
	checkCudaErrors(hipFree(dist));
    checkCudaErrors(hipFree(sigma));
	checkCudaErrors(hipFree(delta));
	checkCudaErrors(hipFree(P));
    hipDeviceReset();
}

