#include "hip/hip_runtime.h"
#include "BC_GPU.cuh"

using namespace std;

__global__ void
betweenness_edge_array_kernel(float *BC, int * r, int * edge_begin, int *edge_end, int * dist, float * sigma, float * delta, bool * P, int numVertices, int numEdges, int offset_source) 
{
	int offset_vertices = blockIdx.x * numVertices;
	int offset_edge = blockIdx.x * numEdges;

	for (int i = threadIdx.x; i < numVertices; i += blockDim.x)
	{
		dist[offset_vertices + i] = -1;
		sigma[offset_vertices + i] = 0;
		delta[offset_vertices + i] = 0;
	}
	for (int i = threadIdx.x; i < numEdges; i += blockDim.x)
	{
		P[offset_edge + i] = false;
	}

	int source = blockIdx.x + offset_source;
	if (source >= numVertices)
		return;
	__shared__ bool done;
	done = false;
	
	int level = 0;
	dist[offset_vertices + source] = level++;
	sigma[offset_vertices + source] = 1; 
	while (!done)
	{
		__syncthreads(); // attention: this sync is neccessary
		done = true;
		for (int edge = threadIdx.x; edge < numEdges; edge += blockDim.x)
		{
			int current = edge_begin[edge];
			if (dist[offset_vertices + current] != level - 1)
				continue;

			int next = edge_end[edge];
			int read_dist = dist[offset_vertices + next];
			if (read_dist == -1)
			{
				dist[offset_vertices + next] = level;
				done = false;
			}

			if (read_dist < level && read_dist >= 0)
				continue;

			atomicAdd(sigma + offset_vertices + next, sigma[offset_vertices + current]); //atomic!
			P[offset_edge + edge] = true;
			
		}
		level ++;
		__syncthreads();
	}
	for (int i = level - 2; i >= 0; i--)
	{
		//// slower method due to atomic operations and bandwidth wastes. 
		//for (int edge = threadIdx.x; edge < numEdges; edge += blockDim.x)
		//{
		//	int current = edge_begin[edge];
		//	if (dist[offset_vertices + current] != i)
		//		continue;
		//	
		//	if (P[offset_edge + edge] != 1)
		//		continue;
		//	int next = edge_end[edge];
		//	atomicAdd(delta + offset_vertices + current, (double) sigma[offset_vertices + current] / sigma[offset_vertices + next]*(1 + delta[offset_vertices + next]));

		//}
		//__syncthreads();

		for (int current = threadIdx.x; current < numVertices; current += blockDim.x)
		{
			if (dist[offset_vertices + current] != i)
				continue;
			for (int j = r[current]; j < r[current + 1]; j += 1)
			{
				if (P[offset_edge + j] != true)
					continue;
				int next = edge_end[j];
				delta[offset_vertices + current] += (double) sigma[offset_vertices + current] / sigma[offset_vertices + next]*(1 + delta[offset_vertices + next]);
			}
		}
		__syncthreads();
	}

	for (int current = threadIdx.x; current < numVertices; current += blockDim.x)
	{
		if(current != source)
			atomicAdd(BC + current, delta[offset_vertices + current]);
	}
}

void Betweenness_GPU_edge_array(int * r, int *r_full, int *c, int numVertices, int numEdges, float *BC, int grid, int thread)
{
    int devID;
    hipDeviceProp_t deviceProps;

    devID = findCudaDevice();

    // get number of SMs on this GPU
    checkCudaErrors(hipGetDeviceProperties(&deviceProps, devID));
    printf("CUDA device [%s] has %d Multi-Processors\n", deviceProps.name, deviceProps.multiProcessorCount);

	//int thread = 256;
	//int grid = 100;
    // allocate device memory
	int* d_r;
	int* d_c;
	int* d_r_full;
	int* dist;
	float* sigma;
	float* delta;
	bool* P;

	checkCudaErrors( hipMalloc( (void**) &d_r, sizeof(int) * numVertices));
	checkCudaErrors( hipMalloc( (void**) &d_r_full, sizeof(int) * numEdges));
	checkCudaErrors( hipMalloc( (void**) &d_c, sizeof(int) * numEdges));

	checkCudaErrors( hipMalloc( (void**) &dist, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &sigma, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &delta, sizeof(int) * numVertices * grid));
	checkCudaErrors( hipMalloc( (void**) &P, sizeof(bool) * numEdges * grid));
    // copy host memory to device
	checkCudaErrors( hipMemcpy( d_r, r, sizeof(int) * numVertices, hipMemcpyHostToDevice) );
    checkCudaErrors( hipMemcpy( d_c, c, sizeof(int) * numEdges, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy( d_r_full, r_full, sizeof(int) * numEdges, hipMemcpyHostToDevice) );

    // allocate device memory for result
    float* d_BC;
    checkCudaErrors( hipMalloc( (void**) &d_BC, sizeof(float) * numVertices));
	checkCudaErrors( hipMemset( d_BC, 0, sizeof(float) * numVertices));

	clock_t kernel_time = 0;

    // execute the kernel
	for (int offset_source = 0; offset_source < numVertices; offset_source += grid)
	{
		clock_t time = clock();
		betweenness_edge_array_kernel<<<grid, thread>>>(d_BC, d_r, d_r_full, d_c, dist, sigma, delta, P, numVertices, numEdges, offset_source);
		// check if kernel execution generated and error
		getLastCudaError("Kernel execution failed");
		hipDeviceSynchronize();
		time = clock() - time;

		cout<<offset_source<<" done. Time = "<<time<<"ms."<<endl;
		kernel_time += time;
	}

	cout<<"total kernel time: "<<kernel_time<<"ms."<<endl;

	// copy result from device to host
	checkCudaErrors(hipMemcpy(BC, d_BC, sizeof(float) * numVertices, hipMemcpyDeviceToHost));

    // cleanup memory
	checkCudaErrors(hipFree(d_r));
    checkCudaErrors(hipFree(d_r_full));
    checkCudaErrors(hipFree(d_c));
	checkCudaErrors(hipFree(d_BC));
	checkCudaErrors(hipFree(dist));
    checkCudaErrors(hipFree(sigma));
	checkCudaErrors(hipFree(delta));
	checkCudaErrors(hipFree(P));
    hipDeviceReset();
}