#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include "hipsparse.h"
#include "hipblas.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
using namespace std;

const int n_thread = 256;
const int ny_thread = 16;
const int nx_thread = 16;

__global__ void cal_k_kernel(long N, int *d_R, float *d_CYC3 )
{		
	int i;
	const int blockid   = blockIdx.x;
	const int threadid = threadIdx.x;
	
	for(i = blockid*blockDim.x+threadid; i<N; i+=blockDim.x*gridDim.x )
	{
		float temp = (float) d_R[i+1];
		temp -= d_R[i];
		if (temp<2)
			d_CYC3[i] = 0;
		else 
			d_CYC3[i] = 1.0/(temp*(temp-1));
		//syncthreads();
	}
}

__global__ void pow3_vector(long N, float *d_vec )
{		
	int i;
	const int blockid   = blockIdx.x;
	const int threadid = threadIdx.x;
	
	for(i = blockid*blockDim.x+threadid; i<N; i+=blockDim.x*gridDim.x )
	{
		double temp = d_vec[i];
		temp = pow(temp,1.0/3);
		d_vec[i] = (float) temp;
		//syncthreads();
	}
}

__global__ void init_block (int *R , int*C, float *V, float *S,int N, int block_size)
{
	//__shared__ R_shared[ny_thread+1];
	int i = blockIdx.x*blockDim.y + threadIdx.y;
	int j = 0;
	//R_shared[threadIdx.y] = R[i]; 
	if (i<block_size)
	{
		float temp = 0;
		int startidx = R[i];
		int endidx = R[i+1];
		for (int k = startidx+threadIdx.x ; k < endidx; k += blockDim.x)
		{
			j = C[k];
			temp = V[k];
			S[i*N + j] = temp;
		}
	}
}

__global__ void calc_cci_block (int N, int size, int *R, int *C, float *V, float *S, float *cci)
{
	__shared__ float a[ny_thread][nx_thread];
	int i = blockIdx.x*blockDim.y + threadIdx.y;
	int j = 0;
	if (i<size)
	{
		float temp = 0;
		for (int k = R[i]+threadIdx.x ; k < R[i+1]; k += blockDim.x)
		{
			j = C[k];
			temp +=  V[k]*S[i*N + j];
		}
		a[threadIdx.y][threadIdx.x] = temp;
		syncthreads();

		for(j = ny_thread/2; j > 0; j/=2)
		{
			if (threadIdx.x < j)  a[threadIdx.y][threadIdx.x]+=a[threadIdx.y][threadIdx.x+j] ;
			syncthreads();
		}
		if(threadIdx.x==0)
			cci[i] = a[threadIdx.y][0];
	}
}



__global__ void dot_vv(long N, float alpha, float *d_a, float *d_b )
{		
	int i;
	const int blockid   = blockIdx.x;
	const int threadid = threadIdx.x;
	
	for(i = blockid*blockDim.x+threadid; i<N; i+=blockDim.x*gridDim.x )
	{
		float temp = d_a[i];
		temp = alpha * temp * d_b[i];
		d_a[i] = (float) temp;
		syncthreads();
	}
}

double Cp_2(int * C, int * R,float * V, float * Cp, int N)
{	

	int count[1];
	hipError_t error;
	error = hipGetDeviceCount(count); 
	if (error != hipSuccess)
	{
		cerr<<"no CUDA device found."<<endl;
		return -1;
	}
	int device[10];
	hipGetDevice(device); 
	hipDeviceProp_t prop[1];
	int best, bestCount = 0;
	for (int i = 0; i < count[0]; i++)
	{
		hipGetDeviceProperties(prop, device[i]);
		if (prop-> multiProcessorCount > bestCount)
		{
			bestCount = prop-> multiProcessorCount;
			best = i;
		}
	}
	hipSetDevice(device[best]);
	hipGetDeviceProperties(prop, device[best]);
	printf("GPU Device %d: \"%s\" with compute capability %d.%d\n", device[best], prop[0].name, prop[0].major, prop[0].minor);

	size_t GM_size = prop->totalGlobalMem;
	

	int edge_num = R[N];
	hipError_t cudaStat; //1,cudaStat2,cudaStat3,cudaStat4,cudaStat5,cudaStat6;
	hipblasStatus_t stat_blas;
	hipblasHandle_t handle_blas;
	hipsparseStatus_t status;
	hipsparseHandle_t handle=0;
	hipsparseMatDescr_t descr=0;
	
	int * dev_C;
	int * dev_R;
	float *dev_V;
	//float *dev_Cp;
	float *dev_CYC3_INV;
	//float * dev_cyc3;
	float * dev_cci;
	
		
	cudaStat = hipMalloc((void**)&dev_R, (N+1)*sizeof(int));
	if (cudaStat != hipSuccess)
	{	cout << "dev_R malloc failed\n";
		return -1;
	}
	cudaStat = hipMalloc((void**)&dev_C, R[N]*sizeof(int));
	if(cudaStat != hipSuccess)
	{	cout << "dev_C malloc failed\n";
		return -1;
	}
	cudaStat = hipMalloc((void**)&dev_V, R[N]*sizeof(int));
	if(cudaStat != hipSuccess)
	{	cout << "dev_V malloc failed\n";
		return -1;
	}

	cudaStat = hipMalloc((void**)&dev_CYC3_INV, N*sizeof(float));
	if(cudaStat != hipSuccess)
	{	cout << "dev_CYC3_INV malloc failed\n";
		return -1;
	}

	/*cudaStat = hipMalloc((void**)&dev_cyc3, N*sizeof(float));	
	if(cudaStat != hipSuccess)
	{	cout << "dev_cyc3 alloc failed\n";
		return -1;
	}*/

	cudaStat = hipMalloc((void**)&dev_cci, N*sizeof(float));
	if(cudaStat != hipSuccess)
	{	cout << "dev_cci alloc failed\n";
		return -1;
	}

	cudaStat = hipMemcpy(dev_R, R,(size_t)((N+1)*sizeof(R[0])),hipMemcpyHostToDevice);
	if(cudaStat != hipSuccess)
	{	cout << "dev_R memcpyh2d failed\n";
		return -1;
	}
	
	cudaStat = hipMemcpy(dev_C, C,(size_t)(R[N]*sizeof(C[0])),hipMemcpyHostToDevice);
	if(cudaStat != hipSuccess)
	{	cout << "dev_C memcpyh2d failed\n";
		return -1;
	}

	cudaStat = hipMemcpy(dev_V, V,(size_t)(R[N]*sizeof(V[0])),hipMemcpyHostToDevice);
	if(cudaStat != hipSuccess)
	{	cout << "dev_V memcpyh2d failed\n";
		return -1;
	}
	/*calc the inverse of CYC3*/

	

	cal_k_kernel<<<96,256>>>((long)N, dev_R, dev_CYC3_INV );

	stat_blas = hipblasCreate(&handle_blas) ;
	if (stat_blas != HIPBLAS_STATUS_SUCCESS)
	{	cout<<"blas failed\n";
		return stat_blas;
	}

	int Vm_idx;
	stat_blas = hipblasIsamax( handle_blas, edge_num, dev_V, 1, &Vm_idx);
	if (stat_blas != HIPBLAS_STATUS_SUCCESS)
	{	cout<<"blas failed\n";
		return stat_blas;
	}

	float alpha;
	cudaStat = hipMemcpy(&alpha, dev_V+Vm_idx-1,(size_t) sizeof(float),hipMemcpyDeviceToHost);
	if(cudaStat != hipSuccess)
	{	cout << "V_max memcpyh2d failed\n";
		return -1;
	}
			
	cout<<"V_max = "<<alpha<<endl;
	
	alpha = 1.0/ (alpha);
	stat_blas = hipblasSscal(handle_blas, edge_num, &alpha, dev_V, 1);
	if (stat_blas != HIPBLAS_STATUS_SUCCESS)
	{	cout<<"scal failed\n";
		return stat_blas;
	}

	pow3_vector <<<96,256>>> (edge_num, dev_V);
	
	/* initialize cusparse library */
	status= hipsparseCreate(&handle);
	if(status != HIPSPARSE_STATUS_SUCCESS) 
	{	cout << "CUSPARSE Library initialization failed\n";
		return -1;
	}

	/* create and setup matrix descriptor */
	status= hipsparseCreateMatDescr(&descr); 
	if(status != HIPSPARSE_STATUS_SUCCESS) 
	{
		cout<<"Matrix descriptor initialization failed";
		return -1;
	} 
	hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
	hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO); 
		
	int block_size;
	block_size = GM_size/N/sizeof(float)/4;
	if (block_size > N)
		block_size = N;
	else if (block_size > 1024)
		block_size -= block_size%1024; 
	else if (block_size>512)
		block_size = 512;
	else if (block_size>32)
		block_size = block_size%32;
	else if (block_size>8)
		block_size = 8;
	else if (block_size>2)
		block_size = 2;
	else block_size = 1;
	
	block_size = 96*32;

	float alpha1 = 1.0;
	float beta = 0;
	float * dev_S;
	cudaStat = hipMalloc((void**)&dev_S, sizeof(float)*N*block_size);
	if(cudaStat != hipSuccess)
	{	cout << "dev_S alloc failed\n";
		return -1;
	}

	float * dev_SS;
	cudaStat = hipMalloc((void**)&dev_SS, sizeof(float)*N*block_size);
	if(cudaStat != hipSuccess)
	{	cout << "dev_SS alloc failed\n";
		return -1;
	}

	dim3 threadnum(ny_thread,nx_thread);
	int blocknum;
	
	int number_blocksize = (N + block_size - 1)/block_size;
	cout<<"block_size : "<<block_size<<"\nnumber of blocks : "<<number_blocksize<<endl;

	for (int i = 0; i < number_blocksize; i++ )
	{
		hipMemset(dev_S, 0, sizeof(float)*N*block_size);
		hipMemset(dev_SS, 0, sizeof(float)*N*block_size);
		
		//cout<<block_size;
		int size = block_size;
		if (i == number_blocksize - 1) size = N - i*block_size;
		blocknum = (size+threadnum.y-1)/threadnum.y;
		init_block<<<blocknum,threadnum>>> (dev_R + i*block_size, dev_C, dev_V, dev_S, N, size);
		
		//float * tmp_test = new float[N*block_size];
		//hipMemcpy(tmp_test , dev_S, sizeof(float)*N*block_size, hipMemcpyDeviceToHost);
		//cout<<endl<<endl;
		

		status = hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
			N, size, N, R[N], &alpha1,
			descr, dev_V, dev_R, dev_C,
			dev_S, N, &beta, dev_SS, N);
		if(status != HIPSPARSE_STATUS_SUCCESS) 
		{
			cout<<"Matrix mm failed";
			return -1;
		} 
		

		/*float * tmp_test1 = new float[N*block_size];
		hipMemcpy(tmp_test1 , dev_SS, sizeof(float)*N*block_size, hipMemcpyDeviceToHost);
		for (int k = 0; k < 100; k++)
			cout<<tmp_test1[k]<<endl;
		cout<<endl;
		for (int k = 4900; k < 5000; k++)
			cout<<tmp_test1[k]<<endl;
		delete[]tmp_test1;
		*/

		calc_cci_block<<<blocknum,threadnum>>>(N, size, dev_R + i*block_size, dev_C, dev_V, dev_SS, dev_cci + i*block_size);
		
		/*float * tmp_test2 = new float[size];
		hipMemcpy(tmp_test2 , dev_cci + i*block_size, sizeof(float)*size, hipMemcpyDeviceToHost);
		cout<<endl;
		for (int k = 0; k < 50; k++)
			cout<<tmp_test2[k]<<endl;
		cout<<endl;
		*/
		/*
		int baseS, nnzS;
		// nnzTotalDevHostPtr points to host memory
		int *dev_SR;
		int *dev_SC;
		float *dev_SV;

		hipsparseMatDescr_t descrS = 0;
		status= hipsparseCreateMatDescr(&descrS); 
		if(status != HIPSPARSE_STATUS_SUCCESS) 
		{
		cout<<"Matrix S descriptor initialization failed";
		return -1;
		} 
		hipsparseSetMatType(descrS,HIPSPARSE_MATRIX_TYPE_GENERAL);
		hipsparseSetMatIndexBase(descrS,HIPSPARSE_INDEX_BASE_ZERO); 

		int *nnzTotalDevHostPtr = &nnzS;
		hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
		hipMalloc((void**)&dev_SR, sizeof(int)*(N+1));

		status = hipsparseXcsrgemmNnz(handle,HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, N, 
		descr, edge_num, dev_R, dev_C,
		descr, edge_num, dev_R, dev_C,
		descrS, dev_SR, nnzTotalDevHostPtr );
		if(status != HIPSPARSE_STATUS_SUCCESS) 
		{
			cout<<"Matrix Xmm failed";
			return -1;
		} 

		if(NULL != nnzTotalDevHostPtr){
			nnzS = *nnzTotalDevHostPtr;
		}
		else{
			hipMemcpy(&nnzS , dev_SR+N, sizeof(int), hipMemcpyDeviceToHost);
			hipMemcpy(&baseS, dev_SR , sizeof(int), hipMemcpyDeviceToHost);
			nnzS -= baseS;
		}
		cout<<nnzS<<endl;
		cudaStat = hipMalloc((void**)&dev_SC, sizeof(int)*nnzS);
		if(cudaStat != hipSuccess)
		{	cout << "dev_SC alloc failed\n";
			return -1;
		}
		cudaStat = hipMalloc((void**)&dev_SV , sizeof(float)*nnzS);
		if(cudaStat != hipSuccess)
		{	cout << "dev_SV alloc failed\n";
			return -1;
		}
		status = hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, N,
		descr, edge_num,dev_V, dev_R, dev_C,
		descr, edge_num,dev_V, dev_R, dev_C,
		descrS, dev_SV, dev_SR, dev_SC);
		if(status != HIPSPARSE_STATUS_SUCCESS) 
		{
			cout<<"Matrix Smm failed";
			return -1;
		} 
		
		int *SR = new int[N+1];
		cudaStat = hipMemcpy(SR , dev_SR, (N+1)*sizeof(int), hipMemcpyDeviceToHost);
		if(cudaStat != hipSuccess)
		{	cout << "dev_SR memcpyd2h failed\n";
			return -1;
		}
		*/
	

	    /*
		for (int ii = i*block_size; ii < i*block_size+size; ii++)
		{	
			/*
			int idx = SR[i];
			int nnz = SR[i+1]-SR[i];
			cudaStat = hipMemset(dev_cyc3, 0, N*sizeof(float));
			if(cudaStat != hipSuccess)
			{	cout << "dev_cyc3 memset failed\n";
				return -1;
			}
			status = hipsparseSsctr(handle,nnz, dev_SV+idx, dev_SC+idx, dev_cyc3, HIPSPARSE_INDEX_BASE_ZERO);
			if(status != HIPSPARSE_STATUS_SUCCESS) 
			{
				cout<<"Vector scatter failed";
				return -1;
			} 
			*/
			/*	
			int idx = R[ii];
			int nnz = R[ii+1]-R[ii];
			hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);
			status = hipsparseSdoti(handle, nnz, dev_V+idx, dev_C+idx, dev_SS+(ii-i*block_size)*N, dev_cci+ii, HIPSPARSE_INDEX_BASE_ZERO);
			if(status != HIPSPARSE_STATUS_SUCCESS) 
			{
				cout<<"Vector doti failed";
				return -1;
			} 
			//cout<<endl<<tmp_test[i];
		}
		*/
	}
	
	
	/*float * tmp_test = new float[N];
	hipMemcpy(tmp_test , dev_cci, sizeof(float)*N, hipMemcpyDeviceToHost);
	for (int k = 0; k < 100; k++)
		cout<<tmp_test[k]<<endl;
	*/
	
	hipFree(dev_S);
	hipFree(dev_SS);
	dot_vv<<<96,256>>> (N, 1.0, dev_cci, dev_CYC3_INV);
	cudaStat = hipMemcpy(Cp , dev_cci, N*sizeof(float), hipMemcpyDeviceToHost);
	if(cudaStat != hipSuccess)
	{	cout << "dev_cci memcpyd2h failed\n";
		return -1;
	}

	float mean_Cp = 0;
	hipblasSasum( handle_blas, N, dev_cci, 1, &mean_Cp);
	
	mean_Cp = mean_Cp/N;
	
	//for (int k = 0; k < 50; k++)
		

	cout<<endl<<mean_Cp<<endl;
	
	//delete []SR;

	status = hipsparseDestroy(handle);
	handle = 0;
	if(status != HIPSPARSE_STATUS_SUCCESS) {
		cout<<"CUSPARSE Library release of resources failed\n";
		return -1;
	}

	stat_blas = hipblasDestroy(handle_blas);
	if (stat_blas != HIPBLAS_STATUS_SUCCESS)
		return -1;
	
	hipFree (dev_C);
	hipFree (dev_R);
	hipFree (dev_V);
	hipFree (dev_cci);
	hipFree (dev_CYC3_INV);

	//hipFree (dev_SR);
	//hipFree (dev_SC);
	//hipFree (dev_SV);
	//hipFree (dev_cyc3);
	

	return ((double) mean_Cp);

}
		
	
	
